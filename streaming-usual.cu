#include "hip/hip_runtime.h"

template<int RO=-1> __global__ __launch_bounds__(LBMconsts::Qn) void  streaming_collision(int ibn) {
  if(threadIdx.x!=0) return;

  /*const int ix = blockIdx.x*CUDAstep::Nb.x + threadIdx.x;
  const int iy = blockIdx.y*CUDAstep::Nb.y + threadIdx.y;
  const int iz = blockIdx.z*CUDAstep::Nb.z + threadIdx.z;*/

  ibn+= blockIdx.x;
  if(ibn>=Nx*Ny*Nz) return;

  const int ix = ibn%Nx;
  const int iy = ibn/Nx%Ny;
  const int iz = ibn/(Nx*Ny);

  const int ild = 0;
  const int ist = 1;

  Cell cell = pars.data.get_cell(ild, ix,iy,iz);
  ftype rho=cell.rho;
  ftype3 vel = cell.vel;

  const int3 ic = make_int3(ix, iy, iz);

  const int3 Nxyz = make_int3(Nx,Ny,Nz);
  using namespace LBMconsts;

  ftype fnew[Qn], feq[Qn];
  ftype4 Vrho = make_ftype4(0,0,0,0);
  ftype T=0;

  for(int iq=0; iq<Qn; iq++) {
    const int3 icn = (ic-e[iq]+Nxyz)%Nxyz;
    const int nind = icn.x + icn.y*Nx + icn.z*Nx*Ny;
    fnew[iq] = pars.data.tiles[ild][nind].f[iq];
    Vrho+= make_ftype4(e[iq].x,e[iq].y,e[iq].z,1)*fnew[iq];
    T+= dot(e[iq],e[iq])*fnew[iq];
  }

  const ftype3 Vel = make_ftype3(Vrho.x,Vrho.y,Vrho.z)/Vrho.w;
  T = T/Vrho.w - dot(Vel,Vel);
  T/= DIM;

  ftype3 Qm=make_ftype3(0,0,0);
  ftype3 Qp=make_ftype3(0,0,0);
  const ftype R=1;
  const ftype lx=1,ly=1,lz=1;
  Cell ncell;
  ncell = pars.data.get_cell(ild, (ix-1+Nx)%Nx,iy,iz); Qm.x = ncell.rho*ncell.vel.x*(lx*lx - 3*R*ncell.T-ncell.vel.x*ncell.vel.x);
  ncell = pars.data.get_cell(ild, (ix+1   )%Nx,iy,iz); Qp.x = ncell.rho*ncell.vel.x*(lx*lx - 3*R*ncell.T-ncell.vel.x*ncell.vel.x);
  ncell = pars.data.get_cell(ild, ix,(iy-1+Ny)%Ny,iz); Qm.y = ncell.rho*ncell.vel.y*(ly*ly - 3*R*ncell.T-ncell.vel.y*ncell.vel.y);
  ncell = pars.data.get_cell(ild, ix,(iy+1   )%Ny,iz); Qp.y = ncell.rho*ncell.vel.y*(ly*ly - 3*R*ncell.T-ncell.vel.y*ncell.vel.y);
  ncell = pars.data.get_cell(ild, ix,iy,(iz-1+Nz)%Nz); Qm.z = ncell.rho*ncell.vel.z*(lz*lz - 3*R*ncell.T-ncell.vel.z*ncell.vel.z);
  ncell = pars.data.get_cell(ild, ix,iy,(iz+1   )%Nz); Qp.z = ncell.rho*ncell.vel.z*(lz*lz - 3*R*ncell.T-ncell.vel.z*ncell.vel.z);

  const ftype3 difQ = 0.5*(Qp-Qm);

  Cell::calcEq(feq, Vrho.w, Vel, T, difQ);
  collision(fnew,feq);

  cell.rho = 0;
  cell.vel = make_ftype3(0,0,0);
  cell.T = 0;
  for(int iq=0; iq<Qn; iq++) {
    cell.f[iq] = fnew[iq];
    cell.rho+= fnew[iq];
    ftype3 ef = make_ftype3(e[iq]);
    cell.vel+= ef*fnew[iq];
    cell.T+= dot(ef,ef)*fnew[iq];
  }
  cell.vel/= cell.rho;
  cell.T = cell.T/cell.rho - dot(cell.vel,cell.vel);
  cell.T/= DIM;

  pars.data.set_cell(cell, ist, ix,iy,iz);
}
