#include "hip/hip_runtime.h"
#include "phys.h"
__device__ inline void Cell::calcEq(ftype feq[Qn], const ftype Rho, const ftype3 Velocity, const ftype Tempr, const ftype3 _difQ={0,0,0}){
  using namespace LBMconsts;
  const ftype rho = Rho;
  ftype3 u = Velocity;
  if(rho==0) u = make_ftype3(0,0,0);
  ftype dT = dcs2;
  ftype Tcur = cs2;
  const ftype T0=cs2;

  const ftype dT2 = dT*dT;
  const ftype dT4 = dT2*dT2;
  
  const ftype u2 = dot(u,u);
  const ftype u4 = u2*u2;
  const int eno = PPdev.EquilibriumOrder;
  const int TERM1 = (eno>=1);
  const int TERM2 = (eno>=2);
  const int TERM3 = (eno>=3);
  const int TERM4 = (eno>=4);
  if(PPdev.IsothermalRelaxation) Tcur=cs2; else Tcur=Tempr;
  const ftype mxwU = 1 - TERM2*u2*0.5*dT;
  for(int i=0; i<Qn; i++) {
    ftype3 eidx = make_ftype3(e[i]);
    const ftype ei2 = dot(eidx,eidx);
    const ftype ei4 = ei2*ei2;
    const ftype eu =  dot(eidx,u);
    const ftype eu2 = eu*eu;
    const ftype eu4 = eu2*eu2;
    ftype mxw  = mxwU +
                 TERM1*eu*dT +
                 TERM2*eu2*0.5*dT2 +
                 TERM2*(Tcur-T0)*0.5*dT*(ei2*dT-DIM)+
                 TERM3*1./6.*eu*dT*( eu2*dT2-3*u2*dT + 3*(Tcur-T0)*dT*(ei2*dT-DIM-2) )+
                 TERM4*1./24.*dT4*( eu4 + 3*T0*T0*u4 - 6*T0*eu2*u2 + 6*(Tcur-T0)*eu2*ei2 + 3*(Tcur-T0)*(Tcur-T0)*ei4
                                    - 6*T0*(Tcur-T0)*(Tcur-T0)*(DIM+2)*ei2 + 3*T0*T0*(Tcur-T0)*(Tcur-T0)*DIM*(DIM+2)
                                    - 6*T0*(Tcur-T0)*u2*ei2 - 6*T0*(Tcur-T0)*(DIM+4)*eu2 + 6*T0*T0*(Tcur-T0)*(DIM+2)*u2
                                  );
    feq[i] = w[i]*rho*mxw;
  }
  #ifdef EXTENDED_EQUILIBRIUM
  #ifndef D3Q27
  #error Extended equilibrium works only for D3Q27 now
  #endif
  const ftype lx=1,ly=1,lz=1;
  const ftype3 dl = make_ftype3(1./lx,1./ly,1./lz);
  const ftype R=1;
  if(PPdev.IsothermalRelaxation) Tcur=cs2; else Tcur=Tempr;
  const ftype T = Tcur;
  ftype3 P = make_ftype3(R*T)+u*u;
  const ftype3 difQ = _difQ;
  const ftype dtau = PPdev.dtau;
  P += (2 - dtau)/(2*rho*dtau) * difQ;
  const ftype3 psi0 = make_ftype3(1)-P*dl*dl;
  const ftype3 psiP = 0.5*( u*dl+P*dl*dl);
  const ftype3 psiM = 0.5*(-u*dl+P*dl*dl);
  const ftype xfactors[3] = {psiM.x,psi0.x,psiP.x};
  const ftype yfactors[3] = {psiM.y,psi0.y,psiP.y};
  const ftype zfactors[3] = {psiM.z,psi0.z,psiP.z};
  for(int i=0; i<Qn; i++) {
    const ftype factor = xfactors[e[i].x+1]*yfactors[e[i].y+1]*zfactors[e[i].z+1];
    feq[i] = rho*factor;
  }
  #endif
}
/*__host__ __device__ inline void Cell::calcEq(ftype feq[Qn], const ftype Rho, const ftype3 Velocity, const ftype Tempr){
  using namespace LBMconsts;
  const ftype rho = Rho;
  ftype3 u = Velocity;
  if(rho==0) u = make_ftype3(0,0,0);
  ftype dT = dcs2;
  ftype Tcur = cs2;
  #ifdef NON_ISOTHERMAL_RELAXATION
  Tcur=Tempr
  #endif

  const ftype dT2 = dT*dT;
  const ftype dT3 = dT*dT*dT;
  const ftype dT4 = dT*dT*dT*dT;
  const ftype dT5 = dT*dT*dT*dT*dT;
  
  const ftype u2 = dot(u,u);
  const ftype u4 = u2*u2;
  const int TERM1 = (EqOrder>=1);
  const int TERM2 = (EqOrder>=2);
  const int TERM3 = (EqOrder>=3);
  const int TERM4 = (EqOrder>=4);
  const int TERM5 = (EqOrder>=5);
  const ftype mxwU = 1 - TERM2*u2*0.5*dT + TERM4*u4*0.125*dT2;
  for(int i=0; i<Qn; i++) {
    ftype3 eidx = make_ftype3(e[i]);
    const ftype eu =  dot(eidx,u);
    const ftype eu2 = eu*eu;
    const ftype eu3 = eu*eu*eu;
    const ftype eu4 = eu*eu*eu*eu;
    const ftype eu5 = eu*eu*eu*eu*eu;
    ftype mxw  = mxwU +
                 TERM1*eu*dT +
                 TERM2*eu2*0.5*dT2 +
                 TERM2*(Tempr-cs2)*0.5*dT*(dot(eidx,eidx)*dT-DIM) +
                 TERM3*eu3*ftype(1./6.)*dT3  - TERM3*eu*u2*0.5*dT2 +
                 TERM4*eu4*ftype(1./24)*dT4  - TERM4*eu2*u2*0.25*dT3 +
                 TERM5*eu5*ftype(1./120)*dT5 - TERM5*eu3*u2*ftype(1./12)*dT4 + TERM5*eu*u4*0.125*dT3;
    feq[i] = w[i]*rho*mxw;
    #ifdef NON_ISOTHERMAL_RELAXATION
    feq[i] = w_get(i,Tempr)*rho*mxw;
    #endif
  }
}*/

inline __host__ __device__ void Data_t::set_cell(const Cell& c, const int ipar, const int ix, const int iy, const int iz){
  static_assert(Tile::Ns==1);
  for(int iq=0; iq<Cell::Qn; iq++) {
    const int3 gCrd = make_int3(ix, iy, iz);
    const int gInd =  gCrd.x + gCrd.y*Nx+ gCrd.z*Nx*Ny;

    if(Tile::Ns==1) {
      tiles[ipar][gInd].f[iq] = c.f[iq];
      tiles[ipar][gInd].uT[0] = make_ftype4(c.vel.x, c.vel.y, c.vel.z, c.T);
    } else {
      Tile* ctile = &tiles[ipar][ gCrd.x/Tile::Ns + gCrd.y/Tile::Ns*(Nx/Tile::Ns) + gCrd.z/Tile::Ns*(Nx/Tile::Ns)*(Ny/Tile::Ns) ];
      const int3 intileCrd = gCrd%Tile::Ns;
      const int Ns3 = Tile::Ns*Tile::Ns*Tile::Ns; 
      ctile->f[iq + (intileCrd.x+intileCrd.y*Tile::Ns+intileCrd.z*Tile::Ns*Tile::Ns)*Ns3 ] = c.f[iq];
    }
  }
}
