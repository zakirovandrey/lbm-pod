#include "hip/hip_runtime.h"
//im3D считывает и визуализирует трёхмерные поля,
//получение исходного кода: bzr checkout bzr+ssh://photon/Save/BZR-for-all/lev/im3D
//автор: Вадим Левченко VadimLevchenko@mail.ru
// запуск: ./im3D <имя-файла-массива> [<имя-файла-массива> ...]
//целевой размер массива от 100 до 1500 элементов по каждой координате
//предполагается, что файлы массивов записаны в формате массивов aivlib-а или drp

#include "cuda_math.h"
#include "fpal.h"
#include "im2D.h"
#include "im3D.hpp"

image2D im2D;
image_pars imHost; __constant__ image_pars im;
__constant__ im3D_pars im3D;

float runTime=0.0, SmoothFPS=0.0;
bool recalc_at_once=true, recalc_always=false, save_anim_flag=false, draw_edges_flag=false;
int anim_acc=0, render_type=3;
texture<floatT4im, hipTextureType3D> data3D_tex;
hipArray* data3D_texArray=0;
texture<short2, hipTextureType3D> data3Dsurf_tex;
hipArray* data3Dsurf_texArray=0;
const char* optfName="im3DI.opt";//Имя файла для сохранения опций визуализации
FILE* gpPipe=0;
int sec1Daxis=0;

//#include <string.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/types.h>
//#include <time.h>

#include <malloc.h>

char WinTitle[1024], addTitleStr[5]; int TitleStrInd=0;
const char* baseTitleStr="2"; int baseTitleFlag=1;
int optfid=-1; int im3DIopt_shift=0;

void im3D_pars4save::load_from_file(const char* fn) {
  int optfid=open(fn, O_RDWR|O_CREAT, 0644);
  if(optfid<0) { printf("Не могу открыть файл %s, загрузка наборов опций визуализации невозможна\n", fn); return; }
  int sz,rs;
  rs = read(optfid, &sz, sizeof(sz));
  if(sz<=0 || sz>sizeof(fpal_pars)) { printf("Illegal Drop format\n"); return; }
  rs=read(optfid, &imHost, sz); printf("Load %dB fpal of %ldB", rs, sizeof(fpal_pars));
  rs = read(optfid, &sz, sizeof(sz));
  if(sz<=0 || sz>sizeof(im3D_pars4save)) { printf("Illegal Drop format\n"); return; }
  rs=read(optfid, this, sz); printf(" & %dB im3D of %ldB\n", rs, sizeof(im3D_pars4save));
  close(optfid);
}
char* im3D_pars::reset_title() {
  char* pTitle=WinTitle, TitleStr[20];
  if(baseTitleFlag%3>0) strcpy(TitleStr,baseTitleStr);
  strncpy(TitleStr+strlen(TitleStr),addTitleStr,4);
  if(baseTitleFlag%3==1 && fName) { sprintf(pTitle, "%s ", fName); pTitle += strlen(pTitle); }
  if(baseTitleFlag%3==2 && dfName) { sprintf(pTitle, "%s ", dfName); pTitle += strlen(pTitle); }
  if(strpbrk(TitleStr,"23")) { sprintf(pTitle, "(%dx%dx%d)", Nx,Ny,Nz); pTitle += strlen(pTitle); }
  if(strpbrk(TitleStr,"xyzXYZ")) { sprintf(pTitle, "/(%dx%dx%d)", ix0,iy0,iz0); pTitle += strlen(pTitle); }
  if(strpbrk(TitleStr,"aA\001\023=-+_06789")) { sprintf(pTitle, " %g<f<%g", imHost.fmin,imHost.fmax); pTitle += strlen(pTitle); }
  if(strpbrk(TitleStr,"pP[]|?{}")) { sprintf(pTitle, " pal[%d]:(%g)^%g*%g*%g|%g;", imHost.palID, imHost.pscale, imHost.gamma_pal, imHost.brightness_coff, imHost.max_rgb, imHost.base_val); pTitle += strlen(pTitle); }
  if(strpbrk(TitleStr,"dDjJmM")) { sprintf(pTitle, " D/J/M:%g/%g/%g;", density, opacity, tstep); pTitle += strlen(pTitle); }
#ifdef CALC_TIME_DIAG
extern float calcTime, calcPerf; extern int TimeStep;
  if(strpbrk(TitleStr,"bG")) { sprintf(pTitle, " calc: %.2f sec, %.2fG cells/sec; %d steps;", 1e-3*calcTime, calcPerf, TimeStep); pTitle += strlen(pTitle); }
#endif
  if(strpbrk(TitleStr,"tT\20")) { sprintf(pTitle, " transp: %s,%d", imHost.transparency_discrete_flag?"discr":"mode",imHost.transparency_mode); pTitle += strlen(pTitle); }
  //if(strpbrk(TitleStr,"gG")) { sprintf(pTitle, " ", ); pTitle += strlen(pTitle); }
  //sprintf(WinTitle, " %.1f fps", , ,recalc_always?SmoothFPS:1000./runTime);
  //printf(WinTitle, " render: %.1f fps", , recalc_always?SmoothFPS:1000./runTime);
  return WinTitle;
}
struct RotMatr {
  double v[3][3];
  RotMatr(int c, double phi) {
    phi *= M_PI/180;
    int cp=(c+1)%3, cm=(c+2)%3;
    for(int i=0; i<3; i++) v[i][c] = v[c][i] = 0,0;
    v[c][c] = 1.0;
    v[cp][cp] = v[cm][cm] = cos(phi);
    v[cm][cp] = sin(phi); v[cp][cm] =-v[cm][cp];
  }
  void operator *= (RotMatr& M) {
    double vo[3][3];
    for(int i=0; i<3; i++) for(int j=0; j<3; j++) vo[i][j] = v[i][j];
    for(int i=0; i<3; i++) for(int j=0; j<3; j++) {
      double vn=0.0;
      for(int k=0; k<3; k++) vn += M.v[i][k]*vo[k][j];
      v[i][j] = vn;
    }
  }
};

std::string im3D_pars::getfName() {
  char fN[]="image.__________";
  if(fName) strncpy(fN, fName, sizeof(fN)-1);//[sizeof(fN)-1] = 0;
  if(strrchr(fN,'.')) strrchr(fN,'.')[0] = 0;
  if(strrchr(fN,' ')) strrchr(fN,' ')[0] = 0;
  if(strrchr(fN,'/')) strrchr(fN,'/')[0] = '_';
  return std::string(fN);
}
std::string im3D_pars::getfDropName(const char* ext, int it) {
  char drop_name[1024];
  sprintf(drop_name, "%s/%s_%04d%s", drop_dir, getfName().c_str(), (it>=0)?it:imHost.nFrame, ext);
  return std::string(drop_name);
}

bool im3D_pars::save_png(int it) {
  im2D.out2png(getfDropName(".png", it).c_str());
  imHost.nFpng++; 
  return false;
}
__global__ void save_gp3D();
const int tileSz=16, tilesN=16;

bool im3D_pars::save_gp(int it) {
  std::string png_name=getfDropName(".png", it);
  std::string gp_name=getfDropName(".gp", it);
  im2D.out2png(png_name.c_str());
  //sprintf( gp_name, "a.gp", fName, imHost.nFrame);
  FILE* gp=fopen(gp_name.c_str(), "w"),* old_stdout=stdout;
  fprintf(gp, "unset key\n");
  fprintf(gp, "unset border\n");
  fprintf(gp, "unset xtics\n");
  fprintf(gp, "set x2tics border\n");
  fprintf(gp, "set x2range [%g:%g]\n", imHost.fmin, imHost.fmax);
  fprintf(gp, "unset ytics\n");
  //fprintf(gp, "load \"labels.gp\"\n");
  //printf("viewRotation: %g, %g\n", viewRotation[0], viewRotation[1]);
  //printf("viewTranslation: %g, %g, %g\n", viewTranslation[0], viewTranslation[1], viewTranslation[2]);
  if(render_type==3) {
    const int Sgp=(tilesN-1)*tileSz;
    stdout = gp;
    if(CHECK_ERROR(hipDeviceSynchronize())) throw(-1);
    save_gp3D <<<dim3((im2D.Nx+Sgp-1)/Sgp,(im2D.Ny+Sgp-1)/Sgp),dim3(tilesN,tilesN)>>>();
    if(CHECK_ERROR(hipDeviceSynchronize())) throw(-1);
    stdout = old_stdout;
  }
  fprintf(gp, "plot[0:%g][0:%g] \"%s\" binary filetype=png dx=1 dy=1 with rgbimage\n", float(bNx), float(bNy), png_name.c_str());
  fprintf(gp, "pause -1\n");
  fclose(gp);
  if(type_diag_flag>=0) printf("Зарамочное оформление сохранено в %s\n", gp_name.c_str());
  return false;
}

floatT4im get_val_from_arr3D(int ix, int iy, int iz);
void reset(im3D_pars* p=0);
#if DATA_VECTOR_SZ==1
std::string im3D_pars::save_section(int it) {
  printf("f(%d,%d,%d) = %g\n", ix0, iy0, iz0, get_val_from_arr3D(ix0, iy0, iz0));
  std::string dat_name=getfDropName(".dat",it);
  FILE* dat=fopen(dat_name.c_str(), "w");
  for(int i=0; i<Nx; i++) fprintf(dat, "%d %g\n", i, get_val_from_arr3D(i, iy0, iz0));
  fprintf(dat, "\n\n");
  for(int i=0; i<Ny; i++) fprintf(dat, "%d %g\n", i, get_val_from_arr3D(ix0, i, iz0));
  fprintf(dat, "\n\n");
  for(int i=0; i<Nz; i++) fprintf(dat, "%d %g\n", i, get_val_from_arr3D(ix0, iy0, i));
  fclose(dat);
  return dat_name;
}
void im3D_pars::plot_section() {
  const char* re=gpPipe?"re":"";
  if(gpPipe==NULL) gpPipe = popen("gnuplot", "w");
  int sec[]={ix0,iy0,iz0,ix0,iy0};
  if(sec1Daxis<3) fprintf(gpPipe, "set style data l;\n%splot '%s' i %d t '%c:(%d,%d)'\n", re, save_section().c_str(), sec1Daxis, "xyz"[sec1Daxis], sec[sec1Daxis+1], sec[sec1Daxis+2]);
  else fprintf(gpPipe, "set style data l;\n%splot '%s' u ($1-%d):2 i 0 t '(ix-%d)', '' u ($1-%d):2 i 1 t '(iy-%d)', '' u ($1-%d):2 i 2 t '(iz-%d)'\n", re, save_section().c_str(), ix0,ix0,iy0,iy0,iz0,iz0);
  fflush(gpPipe);
}
#elif DATA_VECTOR_SZ==3
std::string im3D_pars::save_section(int it) {
  floatT4im v=get_val_from_arr3D(ix0, iy0, iz0);
  printf("f(%d,%d,%d) = (%g,%g,%g,%g)\n", ix0, iy0, iz0, v.x, v.y, v.z, v.w);
  std::string dat_name=getfDropName(".dat",it);
  FILE* dat=fopen(dat_name.c_str(), "w");
  for(int i=0; i<Nx; i++) {
    floatT4im v=get_val_from_arr3D(i, iy0, iz0);
    fprintf(dat, "%d %g %g %g %g\n", i, v.x, v.y, v.z, v.w);
  }
  fprintf(dat, "\n\n");
  for(int i=0; i<Ny; i++) {
    floatT4im v=get_val_from_arr3D(ix0, i, iz0);
    fprintf(dat, "%d %g %g %g %g\n", i, v.x, v.y, v.z, v.w);
  }
  fprintf(dat, "\n\n");
  for(int i=0; i<Nz; i++) {
    floatT4im v=get_val_from_arr3D(ix0, iy0, i);
    fprintf(dat, "%d %g %g %g %g\n", i, v.x, v.y, v.z, v.w);
  }
  fclose(dat);
  return dat_name;
}
void im3D_pars::plot_section() {
  const char* re=gpPipe?"re":(render_type==3?"s":"");
  if(gpPipe==NULL) gpPipe = popen("gnuplot", "w");
  int sec[]={ix0,iy0,iz0,ix0,iy0};
  if(render_type==3) {
    if(sec1Daxis<3) fprintf(gpPipe, "set ticslevel 0; set style data lp;\n%splot '%s' u 2:3:4 i %d t '%c:(%d,%d)'\n", re, save_section().c_str(), sec1Daxis, "xyz"[sec1Daxis], sec[sec1Daxis+1], sec[sec1Daxis+2]);
    else fprintf(gpPipe, "set ticslevel 0; set style data lp;\n%splot '%s' u 2:3:4 i 0 t '(ix-%d)', '' u 2:3:4 i 1 t '(iy-%d)', '' u 2:3:4 i 2 t '(iz-%d)'\n", re, save_section().c_str(), ix0,iy0,iz0);
  } else if(render_type==2) {
    if(sec1Daxis<3) fprintf(gpPipe, "set style data l;\n%splot '%s' u 1:2 i %d t '[%d,%d].x', '' u 1:3 i %d t '.y', '' u 1:4 i %d t '.z'\n", re, save_section().c_str(), sec1Daxis, sec[sec1Daxis+1], sec[sec1Daxis+2], sec1Daxis, sec1Daxis);
    else fprintf(gpPipe, "set style data l;\n%splot '%s' u ($1-%d):2 i 0 t '(ix-%d).x', '' u ($1-%d):3 i 0 t '.y', '' u ($1-%d):4 i 0 t '.z', '' u ($1-%d):2 i 1 t '(iy-%d).x', '' u ($1-%d):3 i 1 t '.y', '' u ($1-%d):4 i 1 t '.z', '' u ($1-%d):2 i 2 t '(iz-%d).x', '' u ($1-%d):3 i 2 t '.y', '' u ($1-%d):4 i 2 t '.z'\n", re, save_section().c_str(), ix0,ix0,ix0,ix0,iy0,iy0,iy0,iy0,iz0,iz0,iz0,iz0);
  //else fprintf(gpPipe, "set style data l;\n%splot '%s' u ($1-%d):2 i 0 t '(ix-%d)', '' u ($1-%d):2 i 1 t '(iy-%d)', '' u ($1-%d):2 i 2 t '(iz-%d)'\n", re, save_section().c_str(), ix0,ix0,iy0,iy0,iz0,iz0);
  }
  fflush(gpPipe);
}
#endif

void im3D_pars::clear4exit() {
  im2D.clear();
  CHECK_ERROR(hipFreeArray(data3D_texArray));
  CHECK_ERROR(hipFreeArray(data3Dsurf_texArray));
  CHECK_ERROR(hipFree(randArr));
}
void save_bmp4backgrownd();
any_idle_func_struct xyz_void,* xyz=&xyz_void;
struct idle_func_struct3D: public any_idle_func_struct {
  float* par, val;
  void set(float* _par, float _val) { par = _par; val = _val; }
  void step() { *par += val; }
} xyz3D;
struct idle_func_struct2D: public any_idle_func_struct {
  int* i0, N, di;
  void set(int* _i0, int _N, int _di) { i0=_i0; N=_N; di=_di; }
  void step() { *i0 += di; if(*i0<0) *i0=N-1; else if(*i0>=N) *i0=0; }
} xyz2D;
idle_func_calc icalc;
template<class Tflt>
struct idle_func_calcNdrop: public idle_func_calc {
  FILE* sensorsStr;
  int* sensors;
  int Nsensors;
  idle_func_calcNdrop(): Nsensors(0), sensors(0), sensorsStr(0) {}
  ~idle_func_calcNdrop() { delete sensors; }
  void add_sensor(int ix, int iy, int iz) {
    int* pi=sensors;
    for(int i=0; i<Nsensors; i++, pi+=3) if(pi[0] == ix && pi[1] == iy && pi[2] == iz)
      { printf("Сенсор (%d,%d,%d) уже задан. Вы делаете что-то не то!\n", ix, iy, iz); return; }
    Nsensors++;
    printf("Создаю новый сенсор в точке (%d,%d,%d), теперь их %d, файл <sensors.dat> будет очищен.\n", ix, iy, iz, Nsensors);
    if(sensors == 0) sensors = (int*)malloc(Nsensors*3*sizeof(int));
    else sensors = (int*)realloc(sensors, Nsensors*3*sizeof(int));
    pi = sensors+3*(Nsensors-1);
    pi[0] = ix; pi[1] = iy; pi[2] = iz;
    sensorsStr = fopen("sensors.dat", "w");
    pi=sensors;
    fprintf(sensorsStr, "#");
    for(int i=0; i<Nsensors; i++, pi+=3) fprintf(sensorsStr, "\t(%d,%d,%d)", pi[0],pi[1],pi[2]);
    fprintf(sensorsStr, "\n");
    fclose(sensorsStr);
  }
#if DATA_VECTOR_SZ==1
void step() {
  idle_func_calc::step();
  if(Nsensors==0) return;
  sensorsStr = fopen("sensors.dat", "a");
  fprintf(sensorsStr, "%g", t);
  int* pi=sensors;
  for(int i=0; i<Nsensors; i++, pi+=3) fprintf(sensorsStr, "\t%g", get_val_from_arr3D(pi[0], pi[1], pi[2]));
  fprintf(sensorsStr, "\n");
  fclose(sensorsStr);
}
void plot_sensors() {
  if(Nsensors==0) return;
  if(gpPipe==NULL) gpPipe = popen("gnuplot", "w");
  int* pi=sensors;
  fprintf(gpPipe, "set style data l;\nplot 'sensors.dat' u 1:2 t '%d,%d,%d'", pi[0],pi[1],pi[2]); pi+=3;
  for(int i=1; i<Nsensors; i++, pi+=3) fprintf(gpPipe, ", '' u 1:%d t '%d,%d,%d'", i+2, pi[0],pi[1],pi[2]);
  fprintf(gpPipe, "\n");
  fflush(gpPipe);
}
#elif DATA_VECTOR_SZ==3
void step() {
  idle_func_calc::step();
  if(Nsensors==0) return;
  sensorsStr = fopen("sensors.dat", "a");
  fprintf(sensorsStr, "%g", t);
  int* pi=sensors;
  for(int i=0; i<Nsensors; i++, pi+=3) {
    floatT4im v=get_val_from_arr3D(pi[0], pi[1], pi[2]);
    fprintf(sensorsStr, "\t%g\t%g\t%g", v.x, v.y, v.z);
  }
  fprintf(sensorsStr, "\n");
  fclose(sensorsStr);
}
void plot_sensors() {
  if(Nsensors==0) return;
  if(gpPipe==NULL) gpPipe = popen("gnuplot", "w");
  int* pi=sensors;
  if(render_type==3) {
    fprintf(gpPipe, "set style data lp; set ticslevel 0;\nsplot 'sensors.dat' u 2:3:4 t '%d,%d,%d'", pi[0],pi[1],pi[2]); pi+=3;
    for(int i=1; i<Nsensors; i++, pi+=3) fprintf(gpPipe, ", '' u %d:%d:%d t '%d,%d,%d'", 3*i+2,3*i+3,3*i+4, pi[0],pi[1],pi[2]);
  } else if(render_type==2) {
    fprintf(gpPipe, "set style data l;\nplot 'sensors.dat' u 1:2 t '[%d,%d,%d].x', '' u 1:3 t '[%d,%d,%d].y', '' u 1:4 t '[%d,%d,%d].z'", pi[0],pi[1],pi[2], pi[0],pi[1],pi[2], pi[0],pi[1],pi[2]); pi+=3;
    for(int i=1; i<Nsensors; i++, pi+=3) fprintf(gpPipe, ", '' u 1:%d t '[%d,%d,%d].x', '' u 1:%d t '[%d,%d,%d].y', '' u 1:%d t '[%d,%d,%d].z'", 3*i+2, pi[0],pi[1],pi[2],3*i+3, pi[0],pi[1],pi[2],3*i+4, pi[0],pi[1],pi[2]);
  }
  fprintf(gpPipe, "\n");
  fflush(gpPipe);
}
#endif
};
idle_func_calcNdrop<floatT4im> icalcNdrop;
//void add_sensor(int ix, int iy, int iz) { icalcNdrop.add_sensor(ix, iy, iz); }

#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>

__global__ void init_rand(hiprandState *states, float* randArr) {
  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  hiprand_init(1234, tid, 0, &states[tid]);  //  Initialize CURAND
  randArr[tid] = 2.*M_PI*hiprand_uniform (&states[tid]);     // between 0 and 1
}
__device__ float get_float4lim(float v) { return v; }
__device__ float get_float4lim(float2 v) { return length(v); }
__device__ float get_float4lim(float4 v) { return length(v); }
__global__ void calc_limits3D(uint3 IB, uint3 IE, uint3 blkSz, uint3 Nthr, float2* fLims) {
  float2 fLim;
  IB+=blkSz*blockIdx*make_uint3(blockDim)+make_uint3(threadIdx.x/(Nthr.y*Nthr.z), (threadIdx.x/Nthr.z)%Nthr.y, threadIdx.x%Nthr.z);
  IE=min(IE,IB+blkSz);
  //if(threadIdx.x==0) printf("Blk %d from (%d,%d,%d) to (%d,%d,%d)\n",blockIdx.x+gridDim.x*(blockIdx.y+gridDim.y*blockIdx.z),IB.x,IB.y,IB.z, IE.x,IE.y,IE.z);
  fLim.x = fLim.y = get_float4lim(tex3D(data3D_tex, IB.x,IB.y,IB.z));
  for(int ix=IB.x; ix<IE.x; ix+=Nthr.x) for(int iy=IB.y; iy<IE.y; iy+=Nthr.y) for(int iz=IB.z; iz<IE.z; iz+=Nthr.z) {
    float v=get_float4lim(tex3D(data3D_tex, ix,iy,iz));
    if(v<fLim.x) fLim.x = v;
    if(v>fLim.y) fLim.y = v;
  }
  __shared__ float2 fLim_sh[512];
  fLim_sh[threadIdx.x] = fLim;
  __syncthreads();
  if(threadIdx.x >= warpSize) return;
  for(int i=threadIdx.x; i<blockDim.x; i+=warpSize) {
    float2 v=fLim_sh[i];
    if(v.x<fLim.x) fLim.x = v.x;
    if(v.y>fLim.y) fLim.y = v.y;
  }
  fLim_sh[threadIdx.x] = fLim;
  if(threadIdx.x>0) return;
  for(int i=0; i<warpSize; i++) {
    float2 v=fLim_sh[i];
    if(v.x<fLim.x) fLim.x = v.x;
    if(v.y>fLim.y) fLim.y = v.y;
  }
  fLims[blockIdx.x+gridDim.x*(blockIdx.y+gridDim.y*blockIdx.z)] = fLim;
  //printf("Lim (%d,%d,%d) %d => %g %g\n",blockIdx.x,blockIdx.y,blockIdx.z, blockIdx.x+gridDim.x*(blockIdx.y+gridDim.y*blockIdx.z), fLim.x,fLim.y);
}
float2 set_lim_from_tex(uint3 IB, uint3 N) {
  //if(N.x*N.y*N.z<512) { printf("Too small picture\n"); return make_float2(0.,1.); }
  int ind=0; uint3 Ns=N, Nthr;
  if(Ns.x<Ns.y) { ind += 3; int t=Ns.x; Ns.x=Ns.y; Ns.y=t; }
  if(Ns.y<Ns.z) { ind ++; int t=Ns.y; Ns.y=Ns.z; Ns.z=t; }
  if(Ns.x<Ns.y) { ind ++; int t=Ns.x; Ns.x=Ns.y; Ns.y=t; }
  for(Nthr.z=1; Nthr.z<8&&Nthr.z<Ns.z; Nthr.z*=2);
  for(Nthr.y=1; Nthr.y*Nthr.z<64&&Nthr.y<Ns.y; Nthr.y*=2);
  for(Nthr.x=1; Nthr.x*Nthr.y*Nthr.z<512&&Nthr.x<Ns.x; Nthr.x*=2);
  //printf("set Lim from tex: from (%d,%d,%d) size (%d,%d,%d); ind %d; Nthr: (%d,%d,%d)\n",IB.x,IB.y,IB.z, N.x,N.y,N.z, ind, Nthr.x,Nthr.y,Nthr.z);
  if(ind%3==2) { int t=Nthr.x; Nthr.x=Nthr.y; Nthr.y=t; }
  if(ind%3>=1) { int t=Nthr.y; Nthr.y=Nthr.z; Nthr.z=t; }
  if(ind  >=3) { int t=Nthr.x; Nthr.x=Nthr.y; Nthr.y=t; }

  uint3 Sblk=make_uint3(512), Nblk=(N+(Sblk-1))/Sblk;
  int NNblk=Nblk.x*Nblk.y*Nblk.z;
  float2 fLim,* fLims=0,* fLimsD=0;
  if(CHECK_ERROR(hipMalloc((void**) &fLimsD, NNblk*sizeof(float2)))) throw(-1);
  //printf("Lim: %d*%d*%d => %d Blks, %d %d %d Thrs\n",Nblk.x,Nblk.y,Nblk.z,NNblk, Nthr.x,Nthr.y,Nthr.z);
  calc_limits3D<<<Nblk,Nthr.x*Nthr.y*Nthr.z>>>(IB, IB+N, Sblk, Nthr, fLimsD);
  fLims=new float2[NNblk];
  if(CHECK_ERROR(hipMemcpy(fLims, fLimsD, NNblk*sizeof(float2), hipMemcpyDeviceToHost))) throw(-1);
  CHECK_ERROR(hipFree(fLimsD));
  fLim = *fLims;
  for(int i=1; i<NNblk; i++) {
    if(fLims[i].x<fLim.x) fLim.x = fLims[i].x;
    if(fLims[i].y>fLim.y) fLim.y = fLims[i].y;
  }
  delete fLims;
  return fLim;
}

int print_help();

void im3D_pars::print_help() {
  ::print_help();
  printf("\
======= Общее управление программой:\n\
 «ESC» \tВыход из программы\n\
  3¦2  \tпереключает рендеринг 3D¦2D в сечениях (%dD)\n\
   4   \tв режиме 3D переключает режим визуализации потенциал/градиентный режим/на поверхности\n\
<Enter¦BackSpace>\tПереход к следующему¦предыдущему массиву\n\
  w¦W  \tСохранение текущего набора опций визуализации в файл «%s»¦то же, но предыдущий набор не переписывается, можно сохранить произвольное число наборов последовательно\n\
  r¦R  \tЗагрузка ранее сохранённых наборов опций последовательно¦загрузка без перехода к следующему набору\n\
«Ctr-r»\tСброс параметров в значения по умолчанию\n\
  f¦F  \tПереход к началу¦концу файла сохранённых наборов опций\n\
  v¦V  \tУвеличение¦уменьшение уровня вывода диагностики (%d)\n\
«Ctr-v»\tПечатает диагностику, особено актуально, если заголовок окна не виден\n\
«Ctr-w»\tПереключает режим вывода в заголовок окна диагностики по умолчанию\n\
  s¦S  \tСохранение картинки в формате png|вместе с зарамочным оформлением в gnuplot\n\
   ~   \tВключает показ положения выделенной точки (x0,y0,z0), xyz при этом работают в режиме 2D\n\
 #¦$¦%% \tПереключение режима зарамочного оформления режима 3D: сетка¦рёбра бокса¦передний план\n\
   @   \tПереключает режим фона: однотонный/сохранённая картинка/2D сечения через выделенную точку/через сетку 3D\n\
   !   \tСохраняет картинку для фона\n\
«Ctr-z»\tУстанавливает координаты точки, относительно которой происходит вращение, в значение выделенной, при этом сдвигается сетка 3D\n\
  k¦K  \tУменьшение¦увеличение ширины линий контура\n\
«Ctr-k»\tВключает режим прорисовки линий контура (в 2D)\n\
  m¦M  \tУменьшение¦увеличение шага вдоль луча для соответствующего изменения точности (%g), ВНИМАНИЕ: при мелком шаге может очень медленно прорисовывать\n\
  e¦E  \tРазмазывание луча по горизонтали для соответствующего изменения муара (%g),\n\
  d¦D  \tУвеличение¦уменьшение плотности цвета при суммировании вдоль луча (%g)\n\
  j¦J  \tУменьшение¦увеличение порога цветовой плотности (%g)\n\
«Ctr-f»\tПереключает режим интерполяции в текстуре с режима по умолчанию (линейный в 3D/point в 2D)\n\
«Ctr-d»\tв 3D режиме отсекает часть массива\n\
«Ctr-L»\tЧитает параметры командной строки из текстового файла <add.opt>, формат: 1 параметр на строку, список значений без кавычек\n\
  a¦A  \tУстановка пределов палитры из пределов текущего массива ¦ из значений fMin..fMax\n\
«Ctr-a»\tУстановка значений fMin..fMax из текущих пределов палитры\n\
«Ctr-s»\tУстановка пределов палитры, используя пределы массива в сечении поперёк выбранной оси\n\
   1   \tпереключает (циклически, по xyz) ось, вдоль которой строится одномерный график в gnuplot  (%c)\n\
o¦«Ctr-o»\tВыводит в окно gnuplot сечение вдоль выбранной оси¦то же с перерисовкой\n\
  O¦Q  \tДля точки (x0,y0,z0): Печатает в терминале значение текущего поля и выводит в файл сечения вдоль лучей, проходящих через неё¦Добавляет сенсор\n\
q¦«Ctr-q»\tсохраняет значения сенсоров в файле sensors.dat¦выводит в окно gnuplot запись сенсоров\n\
======= Управление динамикой:\n\
  g¦G  \tОтключение¦включение постоянной перерисовки в цикле GLUT (%d)\n\
xyz¦XYZ\tВ режиме 2D, а также в 3D в режиме визуализации положения сечений: Увеличение¦уменьшение координат выделенной точки параллелепипеда данных (%d,%d,%d)\n\
xyz¦XYZ\tВ режиме 3D: Вращение вокруг осей x,y,z вперёд¦назад (%g,%g,%g)\n\
  u¦U  \tВ режиме 3D: Приближение¦удаление объекта (%g)\n\
======= Управление мышью (L¦R¦M --- левая¦правая¦средняя кнопки):\n\
   L   \tВ режиме 2D переустанавливает срезы, исходя из координат выбранной точки\n\
 L¦R¦M \tВ режиме 3D: вращение¦изменение масштаба¦сдвиг рисунка\n\
«Ctr-L»\tСдвиг картинки под курсором\n\
 В районе палитры (верхние 20 точек):\n\
 L¦R¦M \tустанавливает нижний¦верхний пределы¦центр палитры, исходя из x-координаты выбранной точки\n\
  L¦R  \tВ режиме «Ctl-t» (бинарной прозрачности) делает цвет прозрачным¦видимым\n\
", render_type, optfName, type_diag_flag, tstep, randR, density, opacity, "xyz"[sec1Daxis], recalc_always, ix0, iy0, iz0, viewRotation[0], viewRotation[1], viewRotation[2], viewTranslation[2]);
  imHost.print_help();
}
// normal          shift           Ctrl
//«DEL»
//`     67            %^&*      `1234567890 
//       i              I         e  yu   []
//         ;'         H   :"      d ghj  ;'\
//     n ,.          BN <>      zx  bnm,./ 
bool im3D_pars::key_func(unsigned char key, int x, int y) {
  recalc_at_once=true;
  size_t rs=0;
  if(key != addTitleStr[TitleStrInd]) addTitleStr[(TitleStrInd++)%4] = key;
  switch(key) {
  case 'A': imHost.set_lim(fMin, fMax); return true;
  case 'a': { float2 fLim=set_lim_from_tex(make_uint3(0,0,0), make_uint3(Nx,Ny,Nz)); imHost.set_lim(fLim.x, fLim.y); } return true;
//  case 'a': { float2 fLim=make_float2(-0.15,+0.15); imHost.set_lim(fLim.x, fLim.y); } return true;
  case 1: { fMin = imHost.fmin; fMax = imHost.fmax; } return true;
  case 19: { float2 fLim=make_float2(-1,1);
    switch(sec1Daxis) {
      case 0: fLim=set_lim_from_tex(make_uint3(ix0,0,0), make_uint3(1,Ny,Nz)); break;
      case 1: fLim=set_lim_from_tex(make_uint3(0,iy0,0), make_uint3(Nx,1,Nz)); break;
      case 2: fLim=set_lim_from_tex(make_uint3(0,0,iz0), make_uint3(Nx,Ny,1)); break;
    }; imHost.set_lim(fLim.x, fLim.y);
  } return true;
  case 18: ::reset(this); return true;
  case 'w': {
    printf("Drop %ldB fpal & %ldB im3D\n", sizeof(fpal_pars), sizeof(im3D_pars4save));
    if(optfid>=0 && im3DIopt_shift) rs=lseek(optfid,-im3DIopt_shift, SEEK_CUR);
  }
  case 'W': if(optfid>=0) {
    int sz=sizeof(fpal_pars); im3DIopt_shift = 0;
    rs=write(optfid, &sz, sizeof(sz)); im3DIopt_shift += rs;
    rs=write(optfid, &imHost, sz); im3DIopt_shift += rs;
    sz = sizeof(im3D_pars4save);
    rs=write(optfid, &sz, sizeof(sz)); im3DIopt_shift += rs;
    rs=write(optfid, this, sz); im3DIopt_shift += rs;
  } recalc_at_once=false; return true;
  case 'R': if(optfid>=0 && im3DIopt_shift) rs=lseek(optfid,-im3DIopt_shift, SEEK_CUR);
  case 'r': if(optfid>=0) {
    int sz=ld_sz.x;
    im3DIopt_shift = 0;
    if(sz==0) {
      rs = read(optfid, &sz, sizeof(sz));
      if(sz<=0 || sz>sizeof(fpal_pars)) { printf("Illegal Drop format\n"); return true; }
      im3DIopt_shift += rs;
    }
    rs=read(optfid, &imHost, sz); printf("Load %ldB fpal of %ldB", rs, sizeof(fpal_pars)); im3DIopt_shift += rs;
    sz=ld_sz.y;
    if(sz==0) {
      rs = read(optfid, &sz, sizeof(sz));
      if(sz<=0 || sz>sizeof(im3D_pars4save)) { printf("Illegal Drop format\n"); return true; }
      im3DIopt_shift += rs;
    }
    rs=read(optfid, this, sz); printf(" & %ldB im3D of %ldB\n", rs, sizeof(im3D_pars4save)); im3DIopt_shift += rs;
    initTex();
  } return true;
  case 'f': if(optfid>=0) lseek(optfid,0, SEEK_SET); recalc_at_once=false; return true;
  case 'F': if(optfid>=0) lseek(optfid,0, SEEK_END); recalc_at_once=false; return true;
  case 23: baseTitleFlag ++; return true; //recalc_at_once=false;
  case 22: recalc_at_once=false;
    printf("%s\nFrame %d (%.2f/%.2f fps), last run Times: %7.2f msec\n", WinTitle, imHost.nFrame, SmoothFPS, 1000./runTime, runTime);
    return true;
  case 'v': recalc_at_once=false; type_diag_flag++; return true;
  case 'V': recalc_at_once=false; type_diag_flag--; return true;
  case 'S': recalc_at_once=save_gp(); return true;
  case 's': recalc_at_once=save_png(imHost.nFpng); return true;
  case 'e': randR *= sqrt(sqrt(2)); return true;
  case 'E': randR /= sqrt(sqrt(2)); return true;
  case 'm': tstep /= sqrt(sqrt(2)); density /= sqrt(sqrt(2)); return true;
  case 'M': tstep *= sqrt(sqrt(2)); density *= sqrt(sqrt(2)); return true;
  case 'd': density *= sqrt(sqrt(2)); return true;
  case 'D': density /= sqrt(sqrt(2)); return true;
  case 'j': opacity = 1.0 - (1.0-opacity)/sqrt(sqrt(2)); return true;
  case 'J': opacity = 1.0 - (1.0-opacity)*sqrt(sqrt(2)); return true;
  case '@': draw_bmp4backgrownd = (draw_bmp4backgrownd+1)%4; return true;
  case '#': draw_mesh_flag ^= true; return true;
  case '$': draw_box_flag ^= true; return true;
  case '%': draw_fg_flag ^= true; return true;
  case  4 : dissect_box_flag ^= true; return true;
  case '~': draw_sec_xyz_flag ^= true; return true;
  case  6 : filterMode_flag ^= true; initTex(); return true;
  case '!': save_bmp4backgrownd(); return true;
  case '2': render_type=2; initTex(); return true;
  case '3': render_type=3; initTex(); return true;
  case '4': mode3D=(mode3D+1)%3; return true;//grad_mode ^= true; imHost.palDim = 1 + 2*grad_mode; return true;
  case '5': imHost.pal3Daxis = (imHost.pal3Daxis+1)%3; return true;
  case 'g': recalc_always=false; return true;
  case 'G': recalc_always=true; return true;
  case 'Q': recalc_at_once=false; icalcNdrop.add_sensor(ix0, iy0, iz0); return true;
  case 'q': icalcNdrop.step(); return true;
  case 11 : contour_flag ^= true; return true;
  case 'k': contour_width *= 1.2; return true;
  case 'K': contour_width /= 1.2; return true;
  case 12 : {
    FILE* cmd=fopen("add.opt", "r"); if(cmd) {
      char str[1024],* argv[2]; argv[0] = str;
      while(fgets(str, 1024, cmd)) {
        char* c=strchr(str, ' ');
        if(c) {
          if(*c==' ') *c = 0;
          argv[1] = c+1;
        } else argv[1] = str;
        init_from_command_line(argv);
      }
      fclose(cmd);
    }}
    //recalc_at_once=false;
    return true;
  case 17: recalc_at_once=false; icalcNdrop.plot_sensors(); return true;
  case 'O': recalc_at_once=false; save_section(); return true;
  case '1': sec1Daxis = (sec1Daxis+1)%4;
    printf("1D section for gnuplot set to %c\n","xyzA"[sec1Daxis]);
  case 15: recalc_at_once=false;
    if(gpPipe) { pclose(gpPipe); gpPipe = 0; }
  case 'o': recalc_at_once=false; plot_section(); return true;
  case 'b': xyz = &icalcNdrop; return true;
  case 26:
    RotPoint[0] = float(ix0)/Nx;
    RotPoint[1] = float(iy0)/Ny;
    RotPoint[2] = float(iz0)/Nz;
    return true;
  case 'x': case 'X': case 'y': case 'Y': case 'z': case 'Z': case 'u': case 'U':
    if(render_type==2 || draw_sec_xyz_flag) { switch(key) {
        case 'x': xyz2D.set(&ix0, Nx, 1); break;
        case 'X': xyz2D.set(&ix0, Nx,-1); break;
        case 'y': xyz2D.set(&iy0, Ny, 1); break;
        case 'Y': xyz2D.set(&iy0, Ny,-1); break;
        case 'z': xyz2D.set(&iz0, Nz, 1); break;
        case 'Z': xyz2D.set(&iz0, Nz,-1); break;
        default: return true;
      } xyz = &xyz2D; xyz->step();
    } else if(render_type==3) { switch(key) {
        case 'x': xyz3D.set(&viewRotation[0], 0.5f); break;
        case 'X': xyz3D.set(&viewRotation[0],-0.5f); break;
        case 'y': xyz3D.set(&viewRotation[1], 0.5f); break;
        case 'Y': xyz3D.set(&viewRotation[1],-0.5f); break;
        case 'z': xyz3D.set(&viewRotation[2], 0.5f); break;
        case 'Z': xyz3D.set(&viewRotation[2],-0.5f); break;
        case 'u': xyz3D.set(&viewTranslation[2], 0.01f); break;
        case 'U': xyz3D.set(&viewTranslation[2],-0.01f); break;
      } xyz = &xyz3D; xyz->step();
    }
    return true;
  case 27: clear4exit(); exit(0);
  default:
    if(imHost.key_func(key, x, y)) return true;
  }
  recalc_at_once=false;
  if(rs==0) return false;
  return false;
}
struct MKstates {
  int ox, oy;
  int buttonState;
  int modState;
  MKstates(): ox(0),oy(0), buttonState(0),modState(0)  {}
  void correct_screen_coor(int& x, int& y) {
    x -= im2D.xPos;
    y += im2D.yPos-(glutGet(GLUT_WINDOW_HEIGHT)-im2D.Ny);
  }
  void grabState(int button, int state, int x, int y) {
    modState = glutGetModifiers();
    if(state == GLUT_DOWN) buttonState  |= 1<<button;
    else if(state == GLUT_UP) buttonState = 0;
    ox = x;
    oy = y;
  }
} mk_state;

bool im3D_pars::special_func(unsigned char key, int x, int y) {
  mk_state.correct_screen_coor(x,y);
  if(type_diag_flag>=2) printf("special_func, keyN=%d, coors=(%d,%d)\n", key, x, y);
  recalc_at_once=true;
  size_t rs=0;
  if(key != addTitleStr[TitleStrInd]) addTitleStr[(TitleStrInd++)%4] = key;
  int modState = glutGetModifiers(), zoom=10;
  if(modState == GLUT_ACTIVE_CTRL) zoom *= 100;
  if(modState == GLUT_ACTIVE_SHIFT) zoom *= 10;
  if(modState == GLUT_ACTIVE_ALT) zoom /= 10;
  switch(key) {
  case GLUT_KEY_PAGE_UP: im2D.yPos = glutGet(GLUT_WINDOW_HEIGHT)-im2D.Ny; return true;
  case GLUT_KEY_PAGE_DOWN: im2D.yPos = 0; return true;
  case GLUT_KEY_DOWN: im2D.yPos += zoom; if(im2D.yPos>0) im2D.yPos=0; return true;
  case GLUT_KEY_UP: im2D.yPos -= zoom; {
    int yPosMax=glutGet(GLUT_WINDOW_HEIGHT)-im2D.Ny;
    if(im2D.yPos<yPosMax) im2D.yPos = yPosMax;
  } return true;
  case GLUT_KEY_HOME: im2D.xPos = 0; return true;
  case GLUT_KEY_END: im2D.xPos = glutGet(GLUT_WINDOW_WIDTH)-im2D.Nx; return true;
  case GLUT_KEY_LEFT: im2D.xPos += zoom; if(im2D.xPos>0) im2D.xPos=0; return true;
  case GLUT_KEY_RIGHT: im2D.xPos -= zoom; {
    int xPosMax=glutGet(GLUT_WINDOW_WIDTH)-im2D.Nx;
    if(im2D.xPos<xPosMax) im2D.xPos = xPosMax;
  } return true;
  }
  recalc_at_once=false;
  if(rs==0) return false;
  return false;
}
void changeCameraRange(float x, float y){};
void draw_scale(){};
void im3D_pars::mouse_func(int button, int state, int x, int y) {
  if(y<20 && state == GLUT_DOWN && !imHost.draw_flag) {
    changeCameraRange(float(x)/float(bNx), -1);
    return;
  }
  if(y<20 && state == GLUT_UP && !imHost.draw_flag) {
    changeCameraRange(-1, float(x)/float(bNx));
    return;
  }
  mk_state.correct_screen_coor(x,y);
  if(y<20 && state == GLUT_DOWN) {
    if(imHost.transparency_discrete_flag) {
      int ic=floor(0.5+(imHost.pscale)*float(x)/float(bNx));
      switch(button) {
        case 0: imHost.transparency_mode |= (1<<ic); break;
        case 1: imHost.transparency_mode ^= (1<<ic); break;
        case 2: imHost.transparency_mode &= ~(1<<ic); break;
      };
    } else {
    float f=imHost.fmin + x/float(bNx)*(imHost.fmax-imHost.fmin);
    switch(button) {
      case 0: imHost.set_lim(f,imHost.fmax); break; 
      case 2: imHost.set_lim(imHost.fmin,f); break; 
      case 1:
      float df=(f-imHost.fmin)>(imHost.fmax-f)?(f-imHost.fmin):(imHost.fmax-f);
      imHost.set_lim(f-df,f+df); break; 
    };
    if(type_diag_flag>=3) printf("mouse pal: %d,%d, button %d, state %d\n", x,y, button, state);
    recalc_at_once=true;
    }
    return;
  }
  mk_state.grabState(button, state, x,y);
  if(render_type==3) {
    if (state == GLUT_UP) {
      RotMatr R=RotMatr(0,viewRotation[0]), Ry=RotMatr(1,viewRotation[1]), Rz=RotMatr(2,viewRotation[2]), RxT=RotMatr(0,viewRotationTmp[0]), RyT=RotMatr(1,viewRotationTmp[1]);
      R *= Ry; R *= Rz; R *= RxT; R *= RyT;
      /*for(int i=0; i<3; i++) {
        printf("(");
        float s2=0.0;
        for(int j=0; j<3; j++) { printf("\t%g", R.v[i][j]); s2 += R.v[i][j]*R.v[i][j]; }
        printf("); %g\n", s2);
      }*/
      //printf("Mouse: (%g,%g,%g)+(%g,%g) -> ", viewRotation[0], viewRotation[1], viewRotation[2], viewRotationTmp[0], viewRotationTmp[1]);
      double Sy=-R.v[2][0], Cy=sqrt(1.-Sy*Sy), phi[3];
      phi[1] = atan2(Sy,Cy);
      if(Cy>0) {
        double Sx=R.v[2][1]/Cy, Cx=R.v[2][2]/Cy; phi[0]=atan2(Sx,Cx);
        double Sz=R.v[1][0]/Cy, Cz=R.v[0][0]/Cy; phi[2]=atan2(Sz,Cz);
      } else {
        double Cxz=R.v[1][1], Sxz=R.v[0][1]*Sy;
        phi[0]=atan2(Sxz, Cxz); phi[2]=0;
      }
      for(int i=0; i<3; i++) viewRotationTmp[i] = 0;
      for(int i=0; i<3; i++) viewRotation[i] = phi[i]*180.0/M_PI;
      //printf(" (%g,%g,%g)\n", viewRotation[0], viewRotation[1], viewRotation[2]);
    }
  } else {
    if (state == GLUT_DOWN  && mk_state.modState != GLUT_ACTIVE_CTRL) { if(0<=x && x<bNx && 0<=y && y<bNy) reset0(x,bNy-1-y); }
  }
  recalc_at_once=true;
  glutPostRedisplay();
}

void im3D_pars::motion_func(int x, int y) {
  mk_state.correct_screen_coor(x,y);
  if(type_diag_flag>=3) printf("motion func: %d,%d -> %d,%d\n",mk_state.ox,mk_state.oy, x,y);
  if(y<20) {
    return;
  }
  float dx, dy;
  dx = (float)(x - mk_state.ox);
  dy = (float)(y - mk_state.oy);

  if(render_type==2) {
  if(mk_state.modState == GLUT_ACTIVE_CTRL) {
    shift0(mk_state.ox,bNy-1-mk_state.oy, x,bNy-1-y);
  }
  } else {
  if(mk_state.modState == GLUT_ACTIVE_CTRL) {
    eyePoint.x = x;
    eyePoint.y = bNy-y;
  } else {
    if (mk_state.buttonState == 4) // right = zoom
      viewTranslation[2] += dy / 100.0f;
    else if (mk_state.buttonState == 2) { // middle = translate
      viewTranslation[0] += dx / 100.0f;
      viewTranslation[1] -= dy / 100.0f;
    }
    else if (mk_state.buttonState == 1) { // left = rotate
      viewRotationTmp[0] += dy / 5.0f; viewRotationTmp[1] += dx / 5.0f;
    }
  }
  }

  mk_state.ox = x;
  mk_state.oy = y;
  recalc_at_once=true;
  glutPostRedisplay();
}
//int cfX=0, cfY=0;

__global__ void im3Dclear(uchar4 bgk_col) {
  int x=blockIdx.x*blockDim.x+threadIdx.x;
  int y=blockIdx.y*blockDim.y+threadIdx.y;
  if(y<im3D.bNy && x<im3D.bNx) im.bmp[x+y*im3D.bNx] = bgk_col;
}
template<int cx, int cy, int cz>
__global__ void im3Ddraw_any(int sh, int i0) {
  int x1=blockIdx.x*blockDim.x+threadIdx.x, x2=blockIdx.y*blockDim.y+threadIdx.y;
  int p1=sh%im3D.bNx+x1, p2=sh/im3D.bNx+x2;
  if(0>p1 || p1>=im3D.bNx || 0>p2 || p2>=im3D.bNy) return;
  int ix = cx==0?i0:((cx==1?x1:x2)*im3D.x_zoom);
  int iy = cy==0?i0:((cy==1?x1:x2)*im3D.y_zoom);
  int iz = cz==0?i0:((cz==1?x1:x2)*im3D.z_zoom);
  if(ix<im3D.Nx && iy<im3D.Ny && iz<im3D.Nz) {
    uchar4 c=im.get_color(tex3D(data3D_tex, ix,iy,iz));
    if(im3D.draw_sec_xyz_flag && (abs(ix-im3D.ix0)<20 && abs(iy-im3D.iy0)<20 && abs(iz-im3D.iz0)<20) && (cx>0 && ix==im3D.ix0 || cy>0 && iy==im3D.iy0 || cz>0 && iz==im3D.iz0)) c = make_uchar4(255-c.x,255-c.y,255-c.z,c.w);
#if DATA_VECTOR_SZ==1
    if(im3D.contour_flag) {
      for(int i=0; i<im3D.cntr_num; i++) {
        float vp=tex3D(data3D_tex, ix+im3D.contour_width,iy,iz);
        float vm=tex3D(data3D_tex, ix-im3D.contour_width,iy,iz);
        float lv=im3D.cntr_levels[i];
        if(vp != 0 && vm != 0 && (vp>0 ^ vm<0) && (vp>lv ^ vm>lv)) { c = make_uchar4(255-c.x,255-c.y,255-c.z,c.w); continue; }
        vp=tex3D(data3D_tex, ix,iy+im3D.contour_width,iz);
        vm=tex3D(data3D_tex, ix,iy-im3D.contour_width,iz);
        if(vp != 0 && vm != 0 && (vp>0 ^ vm<0) && (vp>lv ^ vm>lv)) { c = make_uchar4(255-c.x,255-c.y,255-c.z,c.w); continue; }
        vp=tex3D(data3D_tex, ix,iy,iz+im3D.contour_width);
        vm=tex3D(data3D_tex, ix,iy,iz-im3D.contour_width);
        if(vp != 0 && vm != 0 && (vp>0 ^ vm<0) && (vp>lv ^ vm>lv)) { c = make_uchar4(255-c.x,255-c.y,255-c.z,c.w); continue; }
      }
      //if((1<v && v<1623/1536.) || (-1>v && v>-1623/1536.)) c = make_uchar4(255-c.x,255-c.y,255-c.z,c.w);
    }
#endif
    im.bmp[sh+x1+x2*im3D.bNx] = c;
  }
  //if(x1==128 && x2==128) printf("res(%d,%d,%d)=%g\n", ix,iy,iz, tex3D(data3D_tex, ix,iy,iz));
}
__global__ void draw_pal() {
  int x=blockIdx.x*blockDim.x+threadIdx.x;
  uchar4 col=im.get_color(im.fmin+(float(x)/im3D.bNx)*(im.fmax-im.fmin));
  uchar4* bmp = im.bmp+im3D.pal_sh;
  for(int y=0; y<20; y++, bmp += im3D.bNx) bmp[x] = col;
}
__global__ void draw_wavelength_pal(){
  int x=blockIdx.x*blockDim.x+threadIdx.x;
  uchar4 col=make_uchar4(0,0,0,255);
  float xpos = float(x)/im3D.bNx;
  uchar4* bmp = im.bmp+im3D.pal_sh; 
  for(int y=0; y<20; y++, bmp += im3D.bNx) {
    if(x%10==0 && y>10) col=make_uchar4(255,255,255,255);
    else                col=make_uchar4(0,0,0,255);
    bmp[x] = col;
  }
  float wRange = im3D.wlRange.y-im3D.wlRange.x;
  float2 cameraRange = im3D.cameraRange;
  if(xpos*wRange>cameraRange.x-im3D.wlRange.x && xpos*wRange<cameraRange.y-im3D.wlRange.x) {
    col=make_uchar4(155,0,0,255);
    bmp = im.bmp+im3D.pal_sh;
    for(int y=0; y<20; y++, bmp += im3D.bNx) bmp[x] = col;
  }
}
__global__ void negate() {
  int x=blockIdx.x*blockDim.x+threadIdx.x;
  uchar4 col=make_uchar4(255,255,255,255);
  uchar4* bmp = im.bmp+x;
  for(int y=0; y<im3D.bNy; y++) bmp[y*im3D.bNx] = col-bmp[y*im3D.bNx];
}
float invViewMatrix[12];
typedef struct {
  float4 m[3];
} float3x4;

//Код 3D рендеринга позаимствован из примеров cuda5.5: 2_Graphics/volumeRender/volumeRender_kernel.cu
__constant__ float3x4 c_invViewMatrix;  // inverse view matrix
struct Ray {
  float3 o;   // origin
  float3 d;   // direction
};

__device__ int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar) {
  // compute intersection of ray with all six bbox planes
  float3 invR = make_float3(1.0f) / (r.d+1e-5);
  float3 tbot = invR * (boxmin - r.o);
  float3 ttop = invR * (boxmax - r.o);

  // re-order intersections to find smallest and largest on each axis
  float3 tmin = fminf(ttop, tbot);
  float3 tmax = fmaxf(ttop, tbot);

  // find the largest tmin and the smallest tmax
  float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
  float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

  *tnear = largest_tmin;
  *tfar = smallest_tmax;
  if(im3D.dissect_box_flag) {
    float3 boxmid=boxmin+make_float3(im3D.BoxFactor[0]*(im3D.ix0+1), im3D.BoxFactor[1]*(im3D.iy0+1), im3D.BoxFactor[2]*(im3D.iz0+1));
    float3 ttopC= invR * (boxmid - r.o);
    float3 tminC = fminf(ttopC, tbot);
    float3 tmaxC = fmaxf(ttopC, tbot);
    float largest_tminC = fmaxf(fmaxf(tminC.x, tminC.y), fmaxf(tminC.x, tminC.z));
    float smallest_tmaxC = fminf(fminf(tmaxC.x, tmaxC.y), fminf(tmaxC.x, tmaxC.z));
    if(smallest_tmaxC > largest_tminC && largest_tmin == largest_tminC) *tnear = smallest_tmaxC;
  }

  return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;
}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

__device__ uchar4 rgbaFloatToInt(float4 rgba, uchar4 bk) {
  float a=rgba.w, da=(1.-a)/255.;
  rgba.x = __saturatef(bk.x*da+a*rgba.x);   // clamp to [0.0, 1.0]
  rgba.y = __saturatef(bk.y*da+a*rgba.y);
  rgba.z = __saturatef(bk.z*da+a*rgba.z);
  rgba.w = __saturatef(rgba.w);
  return make_uchar4((rgba.x*255.f), (rgba.y*255.f), (rgba.z*255.f), (rgba.w*255.f));
}

__device__ uchar4 rgbaFloatToInt(float4 rgba) {
  rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return make_uchar4((rgba.x*255.f), (rgba.y*255.f), (rgba.z*255.f), (rgba.w*255.f));
}
__global__ void draw_pal3D() {
  float x=2.0f*(0.5f+blockIdx.x)/gridDim.x-1.0f, y=2.0f*(0.5f+threadIdx.x)/blockDim.x-1.0f;
  float r2=x*x+y*y;
  if(r2>1.0f) return;
  float r=sqrt(r2), r1=sqrt(1.0f-r2);
  uchar4* bmp = im.bmp+(im3D.pal_sh+im3D.bNx*(int(threadIdx.x)-int(blockDim.x/2))+blockIdx.x);
  bmp[0] = rgbaFloatToInt(im.get_color_for3D(make_float4(x,y,0,1)));
  bmp[blockDim.x] = rgbaFloatToInt(im.get_color_for3D(make_float4(0,y,x,1)));
  bmp[2*blockDim.x] = rgbaFloatToInt(im.get_color_for3D(make_float4(x,0,y,1)));
}

__device__ float smooth(float x) { return __saturatef(1.0f-x*x); } 
//------------------------------
inline __device__ void set_boxMinMax(float3& boxMin, float3& boxMax) {
  float3 boxSize=make_float3(im3D.BoxFactor[0]*im3D.Nx, im3D.BoxFactor[1]*im3D.Ny, im3D.BoxFactor[2]*im3D.Nz);
  //boxMax = 0.5f*boxSize;
  //boxMin =-0.5f*boxSize;
  //boxMax = boxSize;
  float3 cntr=(float3&)im3D.RotPoint*boxSize;
  boxMax = boxSize-cntr;
  boxMin =-cntr;
}
inline __device__ void set_eyeRay(Ray& eyeRay, float x, float y) {
  const float dbNxy=2.0f/(im3D.bNx+im3D.bNy);
  const int Nsum=im3D.Nx+im3D.Ny+im3D.Nz;
  eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 0.32f*Nsum)));
  eyeRay.d = normalize(make_float3((x-im3D.eyePoint.x)*dbNxy, (y-im3D.eyePoint.y)*dbNxy, -2.0f));
  eyeRay.d = mul(c_invViewMatrix, eyeRay.d);
}

__device__ uchar4& get_backgrownd(Ray r, float3 boxmin, float3 boxmax, int bmp_sh) {
  float3 bkgr_col=(float3&)im3D.bkgr_col, box_shrink=(float3&)im3D.box_shrink;
  float3 boxMin=box_shrink*boxmin, boxMax=box_shrink*boxmax;
  float3 fcol=make_float3(0);
  uchar4& vbmp=im.bmp[bmp_sh];
  if(im3D.draw_bmp4backgrownd && im.bmp4backgrownd != 0) vbmp = im.bmp4backgrownd[bmp_sh];
  else { fcol = bkgr_col; vbmp = make_uchar4(0,0,0,0); }
  if(im3D.draw_mesh_flag || im3D.draw_box_flag) {
    float3 invR = make_float3(1.0f) / (r.d+1e-5);
    float3 tB = invR * (boxMin - r.o);
    float3 tT = invR * (boxMax - r.o);

    float tz=r.d.z<0?tB.z:tT.z, xZ=r.o.x+r.d.x*tz, yZ=r.o.y+r.d.y*tz;
    float ty=r.d.y<0?tB.y:tT.y, zY=r.o.z+r.d.z*ty, xY=r.o.x+r.d.x*ty;
    float tx=r.d.x<0?tB.x:tT.x, yX=r.o.y+r.d.y*tx, zX=r.o.z+r.d.z*tx;
    float mval=im3D.Dmesh;
    float3 mb=(float3&)im3D.MeshBox;
    float3 ms=(float3&)im3D.MeshShift;
    if(im3D.draw_box_flag) {
      float xZn=xZ-boxmin.x, yZn=yZ-boxmin.y, xZx=boxmax.x-xZ, yZx=boxmax.y-yZ;
      float zYn=zY-boxmin.z, xYn=xY-boxmin.x, zYx=boxmax.z-zY, xYx=boxmax.x-xY;
      float yXn=yX-boxmin.y, zXn=zX-boxmin.z, yXx=boxmax.y-yX, zXx=boxmax.z-zX;
      float zval=im3D.Dmesh, dm=im3D.Dmesh;
      if(xZn>=-dm && yZn>=-dm && xZx>=-dm && yZx>=-dm) {
        if(im3D.draw_mesh_flag) { mval=fminf(mval,fminf(fabsf(remainderf(xZ-ms.x, mb.x)), fabsf(remainderf(yZ-ms.y, mb.y)))); }
        zval=fminf(zval,fminf(fminf(fabs(xZn), fabs(yZn)), fminf(fabs(xZx), fabs(yZx))));
      }
      if(zYn>=-dm && xYn>=-dm && zYx>=-dm && xYx>=-dm) {
        if(im3D.draw_mesh_flag) { mval=fminf(mval,fminf(fabsf(remainderf(zY-ms.z, mb.z)), fabsf(remainderf(xY-ms.x, mb.x)))); }
        zval=fminf(zval,fminf(fminf(fabs(xYn), fabs(zYn)), fminf(fabs(xYx), fabs(zYx))));
      }
      if(yXn>=-dm && zXn>=-dm && yXx>=-dm && zXx>=-dm) {
        if(im3D.draw_mesh_flag) { mval=fminf(mval,fminf(fabsf(remainderf(yX-ms.y, mb.y)), fabsf(remainderf(zX-ms.z, mb.z)))); }
        zval=fminf(zval,fminf(fminf(fabs(zXn), fabs(yXn)), fminf(fabs(zXx), fabs(yXx))));
      }
      float zdel=smooth(zval/im3D.Dmesh);
      fcol = fcol*(1.0f-zdel)+((float3&)(im3D.box_col))*zdel;
    } else {
           if(xZ>=boxmin.x && yZ>=boxmin.y && xZ<=boxmax.x && yZ<=boxmax.y) mval=fminf(fabsf(remainderf(xZ-ms.x, mb.x)), fabsf(remainderf(yZ-ms.y, mb.y)));
      else if(zY>=boxmin.z && xY>=boxmin.x && zY<=boxmax.z && xY<=boxmax.x) mval=fminf(fabsf(remainderf(zY-ms.z, mb.z)), fabsf(remainderf(xY-ms.x, mb.x)));
      else if(yX>=boxmin.y && zX>=boxmin.z && yX<=boxmax.y && zX<=boxmax.z) mval=fminf(fabsf(remainderf(yX-ms.y, mb.y)), fabsf(remainderf(zX-ms.z, mb.z)));
    }
    if(im3D.draw_mesh_flag) {
      float mdel=smooth(mval/im3D.Dmesh);
      fcol = fcol*(1.0f-mdel)+((float3&)(im3D.mesh_col))*mdel;
    }
  }
  vbmp = vbmp+make_uchar4(__saturatef(fcol.x)*255, __saturatef(fcol.y)*255, __saturatef(fcol.z)*255, 255);
  return vbmp;
}

__device__ uchar4& get_foregrownd(Ray r, float3 boxmin, float3 boxmax, int bmp_sh) {
  float3 box_shrink=(float3&)im3D.box_shrink;
  float3 boxMin=box_shrink*boxmin, boxMax=box_shrink*boxmax;
  float3 fcol=make_float3(0);
  uchar4& vbmp=im.bmp[bmp_sh];
  if(im3D.draw_mesh_flag || im3D.draw_box_flag) {
    float3 invR = make_float3(1.0f) / (r.d+1e-5);
    float3 tB = invR * (boxMin - r.o);
    float3 tT = invR * (boxMax - r.o);

    float tz=r.d.z>0?tB.z:tT.z, xZ=r.o.x+r.d.x*tz, yZ=r.o.y+r.d.y*tz;
    float ty=r.d.y>0?tB.y:tT.y, zY=r.o.z+r.d.z*ty, xY=r.o.x+r.d.x*ty;
    float tx=r.d.x>0?tB.x:tT.x, yX=r.o.y+r.d.y*tx, zX=r.o.z+r.d.z*tx;
    float mval=im3D.Dmesh;
    float3 mb=(float3&)im3D.MeshBox;
    float3 ms=(float3&)im3D.MeshShift;
    if(im3D.draw_box_flag) {
      float xZn=xZ-boxmin.x, yZn=yZ-boxmin.y, xZx=boxmax.x-xZ, yZx=boxmax.y-yZ;
      float zYn=zY-boxmin.z, xYn=xY-boxmin.x, zYx=boxmax.z-zY, xYx=boxmax.x-xY;
      float yXn=yX-boxmin.y, zXn=zX-boxmin.z, yXx=boxmax.y-yX, zXx=boxmax.z-zX;
      float zval=im3D.Dmesh, dm=im3D.Dmesh;
      if(xZn>=-dm && yZn>=-dm && xZx>=-dm && yZx>=-dm) {
        if(im3D.draw_mesh_flag) { mval=fminf(mval,fminf(fabsf(remainderf(xZ-ms.x, mb.x)), fabsf(remainderf(yZ-ms.y, mb.y)))); }
        zval=fminf(zval,fminf(fminf(fabs(xZn), fabs(yZn)), fminf(fabs(xZx), fabs(yZx))));
      }
      if(zYn>=-dm && xYn>=-dm && zYx>=-dm && xYx>=-dm) {
        if(im3D.draw_mesh_flag) { mval=fminf(mval,fminf(fabsf(remainderf(zY-ms.z, mb.z)), fabsf(remainderf(xY-ms.x, mb.x)))); }
        zval=fminf(zval,fminf(fminf(fabs(xYn), fabs(zYn)), fminf(fabs(xYx), fabs(zYx))));
      }
      if(yXn>=-dm && zXn>=-dm && yXx>=-dm && zXx>=-dm) {
        if(im3D.draw_mesh_flag) { mval=fminf(mval,fminf(fabsf(remainderf(yX-ms.y, mb.y)), fabsf(remainderf(zX-ms.z, mb.z)))); }
        zval=fminf(zval,fminf(fminf(fabs(zXn), fabs(yXn)), fminf(fabs(zXx), fabs(yXx))));
      }
      float zdel=smooth(zval/im3D.Dmesh);
      fcol = fcol*(1.0f-zdel)+((float3&)(im3D.box_col))*zdel;
    } else {
           if(xZ>=boxmin.x && yZ>=boxmin.y && xZ<=boxmax.x && yZ<=boxmax.y) mval=fminf(fabsf(remainderf(xZ-ms.x, mb.x)), fabsf(remainderf(yZ-ms.y, mb.y)));
      else if(zY>=boxmin.z && xY>=boxmin.x && zY<=boxmax.z && xY<=boxmax.x) mval=fminf(fabsf(remainderf(zY-ms.z, mb.z)), fabsf(remainderf(xY-ms.x, mb.x)));
      else if(yX>=boxmin.y && zX>=boxmin.z && yX<=boxmax.y && zX<=boxmax.z) mval=fminf(fabsf(remainderf(yX-ms.y, mb.y)), fabsf(remainderf(zX-ms.z, mb.z)));
    }
    if(im3D.draw_mesh_flag) {
      float mdel=smooth(mval/im3D.Dmesh);
      fcol = fcol*(1.0f-mdel)+((float3&)(im3D.mesh_col))*mdel;
    }
  }
  vbmp = vbmp+make_uchar4(__saturatef(fcol.x)*255, __saturatef(fcol.y)*255, __saturatef(fcol.z)*255, 255);
  return vbmp;
}

__device__ void mk_pts(int x, int y, uchar4 col) {
  const int ps=2;
  if(x+1<ps || x+ps>=im3D.bNx || y+1<ps || y+ps>=im3D.bNy) return;
  for(int ix=1-ps; ix<ps; ix++) for(int iy=1-ps; iy<ps; iy++)
    im.bmp[(iy+y)*im3D.bNx + x+ix] = col;
}
__device__ void mk_box(int x, int y, uchar4 col) {
  if(x<0 || x+tileSz>=im3D.bNx || y<0 || y+tileSz>=im3D.bNy) return;
  for(int ix=0; ix<tileSz; ix++) im.bmp[y*im3D.bNx + x+ix] = im.bmp[(tileSz+y)*im3D.bNx + x+ix] = col;
  for(int iy=0; iy<tileSz; iy++) im.bmp[(iy+y)*im3D.bNx + x] = im.bmp[(iy+y)*im3D.bNx + x+tileSz] = col;
}
inline bool __device__ is_inside(float2 pt, float2 p0, float2 px, float2 py) {
  float v1=(p0.x - pt.x) * (px.y - p0.y) - (px.x - p0.x) * (p0.y - pt.y);
  float v2=(px.x - pt.x) * (py.y - px.y) - (py.x - px.x) * (px.y - pt.y);
  float v3=(py.x - pt.x) * (p0.y - py.y) - (p0.x - py.x) * (py.y - pt.y);
  return (v1*v2>=0.0 && v1*v3>=0.0 && v2*v3>=0.0);
}
inline float2 __device__ pt_inside(float2 pt, float2 p0, float2 px, float2 py) {
  float2 res;
  res.x = ((pt.x-p0.x)*(py.y-p0.y)-(pt.y-p0.y)*(py.x-p0.x))/((px.x-p0.x)*(py.y-p0.y)-(px.y-p0.y)*(py.x-p0.x));
  res.y = ((pt.x-p0.x)*(px.y-p0.y)-(pt.y-p0.y)*(px.x-p0.x))/((py.x-p0.x)*(px.y-p0.y)-(py.y-p0.y)*(px.x-p0.x));
  return res;
}
__global__ void save_gp3D() {
  __shared__ float2 fm[3][tilesN][tilesN];//координаты точки в области с сеткой
  __shared__ int hit[tilesN][tilesN];//индекс области попадания луча: 1-z 2-y 4-x 0-молоко
  const int Sgp=(tilesN-1)*tileSz;
  int x=blockIdx.x*Sgp+threadIdx.x*tileSz, y=blockIdx.y*Sgp+threadIdx.y*tileSz;
  float3 boxMin, boxMax; set_boxMinMax(boxMin, boxMax);
  boxMax=((float3&)im3D.box_shrink)*boxMax;
  boxMin=((float3&)im3D.box_shrink)*boxMin;
  Ray r; set_eyeRay(r, x,y);
  float3 invR = make_float3(1.0f) / (r.d+1e-5);
  float3 tB = invR * (boxMin - r.o);
  float3 tT = invR * (boxMax - r.o);
  float tz=r.d.z<0?tB.z:tT.z, xZ=r.o.x+r.d.x*tz, yZ=r.o.y+r.d.y*tz;
  float ty=r.d.y<0?tB.y:tT.y, zY=r.o.z+r.d.z*ty, xY=r.o.x+r.d.x*ty;
  float tx=r.d.x<0?tB.x:tT.x, yX=r.o.y+r.d.y*tx, zX=r.o.z+r.d.z*tx;
  fm[2][threadIdx.x][threadIdx.y] = make_float2(xZ, yZ);
  fm[1][threadIdx.x][threadIdx.y] = make_float2(zY, xY);
  fm[0][threadIdx.x][threadIdx.y] = make_float2(yX, zX);
  if(xZ>=boxMin.x && yZ>=boxMin.y && xZ<=boxMax.x && yZ<=boxMax.y) hit[threadIdx.x][threadIdx.y] = 1; //mk_pts(x,y, red);}
  else if(zY>=boxMin.z && xY>=boxMin.x && zY<=boxMax.z && xY<=boxMax.x) hit[threadIdx.x][threadIdx.y] = 2; //mk_pts(x,y, green);}
  else if(yX>=boxMin.y && zX>=boxMin.z && yX<=boxMax.y && zX<=boxMax.z) hit[threadIdx.x][threadIdx.y] = 4; //mk_pts(x,y, blue);}
  else hit[threadIdx.x][threadIdx.y] = 0;
  __syncthreads();

  int hitA=0, hitM=0;
  if(threadIdx.x<tilesN-1 && threadIdx.y<tilesN-1) {
    for(int i=0;i<2;i++) for(int j=0;j<2;j++) {
      int h=hit[threadIdx.x+i][threadIdx.y+j];
      if(h>0) { hitA++; hitM |= h; }
    }
  }
  int cs=abs(2*hitM-7)/2;
  if(hitA==0 || hitA==4 || cs>=3) return;
  bool is4tick=false, is4bnd=false, is4axis=false;
  is4bnd = hitM==1 || hitM==2 || hitM==4;
  is4axis= hitM==3 || hitM==5 || hitM==6;
  int cp=(cs+1)%3, cm=(cs+2)%3;
  float2 tick_sh={0.0,0.0}, tick2sh={0.0,0.0}; float tick_val;
  const float axis_gap=60., tick_gap=20.;
  float2 pt, spt={0.,0.}; float bMax[]={boxMax.x,boxMax.y,boxMax.z}, bMin[]={boxMin.x,boxMin.y,boxMin.z};
  int labN=(blockIdx.x*(tilesN-1)+threadIdx.x)+gridDim.x*(tilesN-1)*(blockIdx.y*(tilesN-1)+threadIdx.y);
  if(is4axis) {
    float2 p0=fm[cm][threadIdx.x][threadIdx.y], px=fm[cm][threadIdx.x+1][threadIdx.y], py=fm[cm][threadIdx.x][threadIdx.y+1];
    if(fabs(p0.x-bMax[cs])<fabs(p0.x-bMin[cs])) { pt.x = bMax[cs]; spt.x = axis_gap; }
    else { pt.x = bMin[cs]; spt.x = -axis_gap; }
    pt.y = fabs(p0.y-bMax[cp])<fabs(p0.y-bMin[cp])?bMax[cp]:bMin[cp];
    tick_sh = pt_inside(pt, p0,px,py);
    tick2sh = pt_inside(pt+spt, p0,px,py);
    printf("set arrow %d from %g,%g to %g,%g front nohead\n", labN, x+tick_sh.x*tileSz,y+tick_sh.y*tileSz, x+tick2sh.x*tileSz,y+tick2sh.y*tileSz);
    printf("set label %d \"%c\" at %g,%g front center\n", labN, "xyz?"[cs], x+tick2sh.x*tileSz,y+tick2sh.y*tileSz+tick_gap*((tick2sh.y<tick_sh.y)?-1.:1.));
  } else if(is4bnd) {
    float2 fmin,fmax; fmin = fmax = fm[cs][threadIdx.x][threadIdx.y];
    for(int i=0;i<2;i++) for(int j=0;j<2;j++) {
      float2 f = fm[cs][threadIdx.x+i][threadIdx.y+j];
      if(f.x<fmin.x) fmin.x = f.x;
      if(f.y<fmin.y) fmin.y = f.y;
      if(f.x>fmax.x) fmax.x = f.x;
      if(f.y>fmax.y) fmax.y = f.y;
    }
    if(fmin.x<bMin[cp] || fmax.x>bMax[cp]) {// cM = cm;
      int mmin=floorf(fmin.y/im3D.MeshBox[cm]), mmax=floorf(fmax.y/im3D.MeshBox[cm]);
      if(mmin != mmax) is4tick = true;
      pt.x = fmin.x<bMin[cp]?bMin[cp]:bMax[cp]; spt.x = fmin.x<bMin[cp]?-tick_gap:tick_gap;
      pt.y = mmax*im3D.MeshBox[cm];
      tick_val = im3D.base[cm] + pt.y*im3D.step[cm];
    } else if(fmin.y<bMin[cm] || fmax.y>bMax[cm]) {// cM = cp;
      int mmin=floorf(fmin.x/im3D.MeshBox[cp]), mmax=floorf(fmax.x/im3D.MeshBox[cp]);
      if(mmin != mmax) is4tick = true;
      pt.x = mmax*im3D.MeshBox[cp];
      pt.y = fmin.y<bMin[cm]?bMin[cm]:bMax[cm]; spt.y = fmin.y<bMin[cm]?-tick_gap:tick_gap;
      tick_val = im3D.base[cp] + pt.x*im3D.step[cp];
    }
    if(is4tick) {
      float2 p0=fm[cs][threadIdx.x][threadIdx.y], px=fm[cs][threadIdx.x+1][threadIdx.y], py=fm[cs][threadIdx.x][threadIdx.y+1], p1=fm[cs][threadIdx.x+1][threadIdx.y+1];
      if(is_inside(pt, p0,px,py)) {
        tick_sh = pt_inside(pt, p0,px,py);
        tick2sh = pt_inside(pt+spt, p0,px,py);
      } else if(is_inside(pt, p1,py,px)) {
        tick_sh = 1.0-pt_inside(pt, p1,py,px);
        tick2sh = 1.0-pt_inside(pt+spt, p1,py,px);
      } else is4tick = false;
      if(is4tick) printf("set label %d \"%g\" at %g,%g front %s\n", labN, tick_val, x+tick2sh.x*tileSz,y+tick2sh.y*tileSz, (tick2sh.x<tick_sh.x)?"right":"left");
    }
  }
  uchar4 red=make_uchar4(255,0,0,0), green=make_uchar4(0,255,0,0), blue=make_uchar4(0,0,255,0);
  uchar4 ltred=make_uchar4(128,0,0,0), ltgreen=make_uchar4(0,128,0,0), ltblue=make_uchar4(0,0,128,0);
  if(is4axis) {
    mk_box(x,y, red);
    mk_pts(x+tick2sh.x*tileSz,y+tick2sh.y*tileSz, red);
  } else if(is4tick) {
    mk_box(x,y, blue);
    mk_pts(x+tick2sh.x*tileSz,y+tick2sh.y*tileSz, blue);
  } else if(is4bnd) mk_box(x,y, green);
  else mk_box(x,y, ltblue);
}
__global__ void __launch_bounds__(1024,1) grad_render3D() {
#if DATA_VECTOR_SZ==1
  const float opacityThreshold = im3D.opacity;
  const float density=im3D.density, brightness=im.max_rgb;
  float3 boxMin, boxMax; set_boxMinMax(boxMin, boxMax);

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  Ray eyeRay; set_eyeRay(eyeRay, x,y);

  uchar4& vbmp=get_backgrownd(eyeRay, boxMin, boxMax, y*im3D.bNx + x);
  float phi=im3D.randArr[threadIdx.x+threadIdx.y*blockDim.x];
  set_eyeRay(eyeRay, x+im3D.randR*cos(phi),y+im3D.randR*sin(phi));
  float tnear, tfar;
  int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

  if (!hit) return;

  if(tnear < 0.0f) tnear = 0.0f;     // clamp to near plane
  float4 sum = make_float4(0.0f);
  const float3 SzfdBox=make_float3(im3D.Nx,im3D.Ny,im3D.Nz)/(boxMax-boxMin);
  float3 pos_sc = (eyeRay.o + eyeRay.d*tnear-boxMin)*SzfdBox-0.5f;
  const float3 step_sc = (eyeRay.d*im3D.tstep)*SzfdBox;
  for(float t=tnear; t<tfar; t+=im3D.tstep, pos_sc += step_sc) {
    // cross stencil:
    float d=im3D.tstep, dd=im.max_rgb_step*0.5/d;
    float dfdx=dd*(tex3D(data3D_tex, pos_sc.x+d, pos_sc.y, pos_sc.z)-tex3D(data3D_tex, pos_sc.x-d, pos_sc.y, pos_sc.z));
    float dfdy=dd*(tex3D(data3D_tex, pos_sc.x, pos_sc.y+d, pos_sc.z)-tex3D(data3D_tex, pos_sc.x, pos_sc.y-d, pos_sc.z));
    float dfdz=dd*(tex3D(data3D_tex, pos_sc.x, pos_sc.y, pos_sc.z+d)-tex3D(data3D_tex, pos_sc.x, pos_sc.y, pos_sc.z-d));
    float4 col = im.get_color_for3D(make_float4(dfdx,dfdy,dfdz,tex3D(data3D_tex, pos_sc.x, pos_sc.y, pos_sc.z)));
    float w=col.w*density*(1.0f - sum.w); col.w = 1;
    sum += col * w;
    if(sum.w >= opacityThreshold) {
      sum -= col*(sum.w - opacityThreshold);
      break;
    }/*
    col.w *= density;
    col.x *= col.w;
    col.y *= col.w;
    col.z *= col.w;
    sum = sum + col*(1.0f - sum.w);

    if (sum.w > opacityThreshold) break;*/
  }
  sum.x *= brightness; sum.y *= brightness; sum.z *= brightness;
  vbmp = rgbaFloatToInt(sum, vbmp);
  if(im3D.draw_fg_flag) vbmp=get_foregrownd(eyeRay, boxMin, boxMax, y*im3D.bNx + x);
  if(im3D.draw_sec_xyz_flag) {
    if(fabs(pos_sc.x-im3D.ix0)<=0.5|| fabs(pos_sc.y-im3D.iy0)<=0.5|| fabs(pos_sc.z-im3D.iz0)<=0.5) vbmp = make_uchar4(255-vbmp.x,255-vbmp.y,255-vbmp.z,vbmp.w);
  }
#endif
}
__global__ void __launch_bounds__(1024,1) surf_render3D() {
#if DATA_VECTOR_SZ==1
  const float opacityThreshold = im3D.opacity;
  const float density=im3D.density, brightness=im.max_rgb;
  float3 boxMin, boxMax; set_boxMinMax(boxMin, boxMax);

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  Ray eyeRay; set_eyeRay(eyeRay, x,y);

  uchar4& vbmp=get_backgrownd(eyeRay, boxMin, boxMax, y*im3D.bNx + x);
  float phi=im3D.randArr[threadIdx.x+threadIdx.y*blockDim.x];
  set_eyeRay(eyeRay, x+im3D.randR*cos(phi),y+im3D.randR*sin(phi));
  float tnear, tfar;
  int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

  if (!hit) return;

  if(tnear < 0.0f) tnear = 0.0f;     // clamp to near plane
  float4 sum = make_float4(0.0f);
  const float3 SzfdBox=make_float3(im3D.Nx,im3D.Ny,im3D.Nz)/(boxMax-boxMin);
  float3 pos_sc = (eyeRay.o + eyeRay.d*tnear-boxMin)*SzfdBox-0.5f;
  const float3 step_sc = (eyeRay.d*im3D.tstep)*SzfdBox;
  for(float t=tnear; t<tfar; t+=im3D.tstep, pos_sc += step_sc) {
    // cross stencil:
    short2 s2=tex3D(data3Dsurf_tex, pos_sc.x, pos_sc.y, pos_sc.z);
    const short MAX_SHORT=(1<<15)-1; const float dMS=1.0f/MAX_SHORT;
    float3 f={0,0,0};
    if(s2.x!=-MAX_SHORT-1 && s2.y!=-MAX_SHORT-1) {
      f.z = s2.x*dMS; float fxy=sqrt(1-f.z*f.z), phi=s2.y*dMS*M_PI;
      f.y = fxy*sin(phi);
      f.x = fxy*cos(phi);
    }
    float4 col = im.get_color_for3D(make_float4(f.x,f.y,f.z,tex3D(data3D_tex, pos_sc.x, pos_sc.y, pos_sc.z)));
    float w=col.w*density*(1.0f - sum.w); col.w = 1;
    sum += col * w;
    if(sum.w >= opacityThreshold) {
      sum -= col*(sum.w - opacityThreshold);
      break;
    }
  }
  sum.x *= brightness; sum.y *= brightness; sum.z *= brightness;
  vbmp = rgbaFloatToInt(sum, vbmp);
  if(im3D.draw_fg_flag) vbmp=get_foregrownd(eyeRay, boxMin, boxMax, y*im3D.bNx + x);
  if(im3D.draw_sec_xyz_flag) {
    if(fabs(pos_sc.x-im3D.ix0)<=0.5|| fabs(pos_sc.y-im3D.iy0)<=0.5|| fabs(pos_sc.z-im3D.iz0)<=0.5) vbmp = make_uchar4(255-vbmp.x,255-vbmp.y,255-vbmp.z,-vbmp.w);
  }
  //if(x==im3D.bNx/2 && y==im3D.bNy/2) printf("Surf: %f,%f,%f,%f*%f/%f*%f => %d,%d,%d\n", sum.x,sum.y,sum.z,sum.w, last_mul,opacityThreshold, brightness, vbmp.x,vbmp.y,vbmp.z);
#endif
}
__global__ void __launch_bounds__(1024,1) render3D() {
  const float opacityThreshold = im3D.opacity;//0.95f;
  const float density=im3D.density, brightness=im.max_rgb;
  float3 boxMin, boxMax; set_boxMinMax(boxMin, boxMax);

  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  //bool isCnt=blockIdx.x==gridDim.x/2 && blockIdx.y==gridDim.y/2 && threadIdx.x == blockDim.x/2 && threadIdx.y == blockDim.y/2;
  //if ((x >= im3D.bNx) || (y >= im3D.bNy)) return;
  //if(x==0 && y==0) printf("block: %gx%gx%g\n", boxMax.x, boxMax.y, boxMax.z);

  // calculate eye ray in world space
  Ray eyeRay; set_eyeRay(eyeRay, x,y);
  //const int Nsum=im3D.Nx+im3D.Ny+im3D.Nz;
  //const float dbNxy=2.0f/(im3D.bNx+im3D.bNy);
  //eyeRay.o = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 0.32f*Nsum)));
  //eyeRay.d = normalize(make_float3((x-im3D.bNx/2)*dbNxy, (y-im3D.bNy/2)*dbNxy, -2.0f));
  //eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

  uchar4& vbmp=get_backgrownd(eyeRay, boxMin, boxMax, y*im3D.bNx + x);
  float phi=im3D.randArr[threadIdx.x+threadIdx.y*blockDim.x];
  set_eyeRay(eyeRay, x+im3D.randR*cos(phi),y+im3D.randR*sin(phi));
  float tnear, tfar;
  int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

  if (!hit) return;

  if(tnear < 0.0f) tnear = 0.0f;     // clamp to near plane
  //if(tnear+im3D.tstep*Nsum<tfar) tfar = tnear+im3D.tstep*Nsum;
  // march along ray from front to back, accumulating color
  float4 sum = make_float4(0.0f);
  //float3 pos = eyeRay.o + eyeRay.d*tnear;
  //float3 step = eyeRay.d*im3D.tstep;
  const float3 SzfdBox=make_float3(im3D.Nx,im3D.Ny,im3D.Nz)/(boxMax-boxMin);
  float3 pos_sc = (eyeRay.o + eyeRay.d*tnear-boxMin)*SzfdBox-0.5f;
  const float3 step_sc = (eyeRay.d*im3D.tstep)*SzfdBox;
  //const float pscale=im.pscale*0.01f, fscale=100.0f*im.fscale, fmin=0.5f-im.fmin*fscale;
//if(isCnt) printf("I am ray: %f(%f)%f step: %f,%f,%f; pos: %f,%f,%f of %d,%d,%d\n", tnear,im3D.tstep,tfar, step_sc.x,step_sc.y,step_sc.z,  pos_sc.x, pos_sc.y, pos_sc.z, im3D.Nx,im3D.Ny,im3D.Nz);
  for(float t=tnear; t<tfar; t+=im3D.tstep, pos_sc += step_sc) {
    // read from 3D texture
    float4 col = im.get_color_for3D(tex3D(data3D_tex, pos_sc.x, pos_sc.y, pos_sc.z));
    float w=col.w*density*(1.0f - sum.w); col.w = 1;
    sum += col * w;
    if(sum.w >= opacityThreshold) {
      sum -= col*(sum.w - opacityThreshold);
      break;
    }/*
    //float f = tex3D(data3D_tex, pos_sc.x, pos_sc.y, pos_sc.z);
    //float4 col = tex1D(fpal_col_tex, 0.5f + pscale*tex1D(fpal_scale_tex, fmin+f*fscale));
    col.w *= density;

    // "under" operator for back-to-front blending
    //sum = lerp(sum, col, col.w);

    // pre-multiply alpha
    col.x *= col.w;
    col.y *= col.w;
    col.z *= col.w;
    // "over" operator for front-to-back blending
    sum = sum + col*(1.0f - sum.w);

    // exit early if opaque
    if (sum.w > opacityThreshold) break;
   // pos_sc += step_sc;
   */
  }
//if(isCnt) printf("I am ray: %f\n",sum.w);
  sum.x *= brightness; sum.y *= brightness; sum.z *= brightness;
  //sum *= brightness;

  // write output color
  vbmp = rgbaFloatToInt(sum, vbmp);
  if(im3D.draw_fg_flag) vbmp=get_foregrownd(eyeRay, boxMin, boxMax, y*im3D.bNx + x);
  //if(threadIdx.x==0 && threadIdx.y==0) vbmp = make_uchar4(255,255,255,255);
  if(im3D.draw_sec_xyz_flag) {
    if(fabs(pos_sc.x-im3D.ix0)<=0.5|| fabs(pos_sc.y-im3D.iy0)<=0.5|| fabs(pos_sc.z-im3D.iz0)<=0.5) vbmp = make_uchar4(255-vbmp.x,255-vbmp.y,255-vbmp.z,-vbmp.w);
  }
}

void im3D_pars::save_bmp4backgrownd() {
try {
  uchar4* devPtr; size_t size;
  if(CHECK_ERROR(hipGraphicsMapResources(1, &im2D.resource, NULL))) throw(-1);
  if(imHost.negate_flag) negate <<<bNx/NW,NW>>>();
  if(CHECK_ERROR(hipGraphicsResourceGetMappedPointer((void**) &devPtr, &size, im2D.resource))) throw(-1);
  if(imHost.bmp4backgrownd != 0) CHECK_ERROR(hipFree(imHost.bmp4backgrownd));
  if(CHECK_ERROR(hipMalloc((void**) &imHost.bmp4backgrownd, size))) throw(-1);
  if(CHECK_ERROR(hipMemcpy(imHost.bmp4backgrownd, devPtr, size, hipMemcpyDeviceToDevice))) throw(-1);
  im2D.unmapAfterDraw();
} catch(...) {
  printf("save_bmp4backgrownd: Возникла какая-то ошибка.\n");
}
}
void im3D_pars::recalc_sec_im3D() {
try {
  imHost.bmp = im2D.map4draw();
  imHost.bind2draw();
  if(CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(im), &imHost, sizeof(imHost)))) throw(-1);
  if(CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(im3D), this, sizeof(im3D_pars)))) throw(-1);
  int NxZ=Nx/x_zoom, NyZ=Ny/y_zoom, NzZ=Nz/z_zoom;
  int NxB=(NxZ+NW-1)/NW, NyB=(NyZ+NW-1)/NW, NzB=(NzZ+NW-1)/NW;
  unsigned char ub[3];
  for(int i=0; i<3; i++) { ub[i] = bkgr_col[i]<0?0:(bkgr_col[i]>1?255:255.*bkgr_col[i]); }
  im3Dclear <<<dim3(bNx/NW,bNy/NW),dim3(NW,NW)>>>(make_uchar4(ub[0], ub[1], ub[2], 255));
  int shX=0,shY=0;
  for(int ix=int(Nx*RotPoint[0])%int(MeshBox[0]); ix<Nx; ix+=MeshBox[0]) {
    if(shX+NyZ>bNx) { shX=0; shY += NzZ+2; } if(shY+NzZ>bNy) break;
    im3Ddraw_any<0,1,2> <<<dim3(NyB,NzB),dim3(NW,NW)>>>(shX+shY*bNx,ix);
    shX += NyZ+2;
  }// if(shX>0) { shX=0; shY += NzZ+2; }
  for(int iy=int(Ny*RotPoint[1])%int(MeshBox[1]); iy<Ny; iy+=MeshBox[1]) {
    if(shX+NxZ>bNx) { shX=0; shY += NzZ+2; } if(shY+NzZ>bNy) break;
    im3Ddraw_any<1,0,2> <<<dim3(NxB,NzB),dim3(NW,NW)>>>(shX+shY*bNx,iy);
    shX += NxZ+2;
  } if(shX>0) { shX=0; shY += NzZ+2; }
  for(int iz=int(Nz*RotPoint[2])%int(MeshBox[2]); iz<Nz; iz+=MeshBox[2]) {
    //printf("draw xy at iz=%d; (%d,%d) -> (%d,%d)..\n", iz, shX,shY, 0,shY +(NyZ+2), );
    if(shX+NxZ>bNx) { shX=0; shY += NyZ+2; } if(shY+NyZ>bNy) break;
    im3Ddraw_any<1,2,0> <<<dim3(NxB,NyB),dim3(NW,NW)>>>(shX+shY*bNx,iz);
    shX += NxZ+2;
  }// if(shX>0) { shX=0; shY += NyZ+2; }
  if(imHost.draw_flag) draw_pal <<<bNx/NW,NW>>>(); else draw_wavelength_pal <<<bNx/NW,NW>>>();
  if(imHost.negate_flag) negate <<<bNx/NW,NW>>>();
  imHost.nFrame++;
  imHost.unbindAfterDraw();
  im2D.unmapAfterDraw();
} catch(...) {
  printf("recalc_im3D: Возникла какая-то ошибка.\n");
}
}
void im3D_pars::shift0(int x, int y, int x1, int y1) {
  int ix,iy, dx=x1-x, dy=y1-y, sh=dx+dy*bNx;
  if(secType!=1) {
    ix=(x-secXsh%bNx)*z_zoom; iy=(y-secXsh/bNx)*y_zoom;
    if(0<=ix && ix<Nz && 0<=iy && iy<Ny) { if(secXsh%bNx+dx>=0 && secXsh/bNx+dy>=0) secXsh += sh; return; }
  } else {
    ix=(x-secXsh%bNx)*y_zoom; iy=(y-secXsh/bNx)*z_zoom;
    if(0<=ix && ix<Ny && 0<=iy && iy<Nz) { if(secXsh%bNx+dx>=0 && secXsh/bNx+dy>=0) secXsh += sh; return; }
  }
  if(secType<2) {
    ix=(x-secYsh%bNx)*x_zoom; iy=(y-secYsh/bNx)*z_zoom;
    if(0<=ix && ix<Nx && 0<=iy && iy<Nz) { if(secYsh%bNx+dx>=0 && secYsh/bNx+dy>=0) secYsh += sh; return; }
  } else {
    ix=(x-secYsh%bNx)*z_zoom; iy=(y-secYsh/bNx)*x_zoom;
    if(0<=ix && ix<Nz && 0<=iy && iy<Nx) { if(secYsh%bNx+dx>=0 && secYsh/bNx+dy>=0) secYsh += sh; return; }
  }
  ix=(x-secZsh%bNx)*x_zoom; iy=(y-secZsh/bNx)*y_zoom;
  if(0<=ix && ix<Nx && 0<=iy && iy<Ny) { if(secZsh%bNx+dx>=0 && secZsh/bNx+dy>=0) secZsh += sh; return; }
}
void im3D_pars::reset0(int x, int y) {
  int ix,iy;
  ix=(x-secZsh%bNx)*x_zoom; iy=(y-secZsh/bNx)*y_zoom;
  if(0<=ix && ix<Nx && 0<=iy && iy<Ny) { ix0 = ix; iy0 = iy; return; }
  if(secType<2) {
    ix=(x-secYsh%bNx)*x_zoom; iy=(y-secYsh/bNx)*z_zoom;
    if(0<=ix && ix<Nx && 0<=iy && iy<Nz) { ix0 = ix; iz0 = iy; return; }
  } else {
    ix=(x-secYsh%bNx)*z_zoom; iy=(y-secYsh/bNx)*x_zoom;
    if(0<=ix && ix<Nz && 0<=iy && iy<Nx) { iz0 = ix; ix0 = iy; return; }
  }
  if(secType!=1) {
    ix=(x-secXsh%bNx)*z_zoom; iy=(y-secXsh/bNx)*y_zoom;
    if(0<=ix && ix<Nz && 0<=iy && iy<Ny) { iz0 = ix; iy0 = iy; return; }
  } else {
    ix=(x-secXsh%bNx)*y_zoom; iy=(y-secXsh/bNx)*z_zoom;
    if(0<=ix && ix<Ny && 0<=iy && iy<Nz) { iy0 = ix; iz0 = iy; return; }
  }
}
void im3D_pars::recalc_im3D() {
try {
  imHost.bmp = im2D.map4draw();
  imHost.bind2draw();
  if(CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(im), &imHost, sizeof(imHost)))) throw(-1);
  if(CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(im3D), this, sizeof(im3D_pars)))) throw(-1);
  //if(CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeof(float4)*3))) throw(-1);
  //if(CHECK_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferShared))) throw(-1);
  //Pal via Tex
  int NxB=(Nx/x_zoom+NW-1)/NW, NyB=(Ny/y_zoom+NW-1)/NW, NzB=(Nz/z_zoom+NW-1)/NW;
  unsigned char ub[3];
  for(int i=0; i<3; i++) { ub[i] = bkgr_col[i]<0?0:(bkgr_col[i]>1?255:255.*bkgr_col[i]); }
  im3Dclear <<<dim3(bNx/NW,bNy/NW),dim3(NW,NW)>>>(make_uchar4(ub[0], ub[1], ub[2], 255));
  im3Ddraw_any<1,2,0> <<<dim3(NxB,NyB),dim3(NW,NW)>>>(secZsh,iz0);
  if(secType<2) im3Ddraw_any<1,0,2> <<<dim3(NxB,NzB),dim3(NW,NW)>>>(secYsh,iy0);
  else          im3Ddraw_any<2,0,1> <<<dim3(NzB,NxB),dim3(NW,NW)>>>(secYsh,iy0);
  if(secType!=1) im3Ddraw_any<0,2,1> <<<dim3(NzB,NyB),dim3(NW,NW)>>>(secXsh,ix0);
  else           im3Ddraw_any<0,1,2> <<<dim3(NyB,NzB),dim3(NW,NW)>>>(secXsh,ix0);
  if(imHost.draw_flag) draw_pal <<<bNx/NW,NW>>>(); else draw_wavelength_pal <<<bNx/NW,NW>>>();
  if(imHost.negate_flag) negate <<<bNx/NW,NW>>>();
  imHost.nFrame++;
  imHost.unbindAfterDraw();
  im2D.unmapAfterDraw();
} catch(...) {
  printf("recalc_im3D: Возникла какая-то ошибка.\n");
}
}
void im3D_pars::recalc3D_im3D() {
try {
  // use OpenGL to build view matrix
  GLfloat modelView[16];
  glMatrixMode(GL_MODELVIEW);
  glPushMatrix();
  glLoadIdentity();
  switch(mk_state.modState) {
    case GLUT_ACTIVE_SHIFT:
  glRotatef(-viewRotation[1], 0.0, 1.0, 0.0);
  glRotatef(-viewRotation[0], 1.0, 0.0, 0.0);
    break;
    case GLUT_ACTIVE_CTRL:
    default:
  glRotatef(-viewRotation[0], 1.0, 0.0, 0.0);
  glRotatef(-viewRotation[1], 0.0, 1.0, 0.0);
  glRotatef(-viewRotation[2], 0.0, 0.0, 1.0);
  glRotatef(-viewRotationTmp[0], 1.0, 0.0, 0.0);
  glRotatef(-viewRotationTmp[1], 0.0, 1.0, 0.0);
  }
  glTranslatef(-viewTranslation[0], -viewTranslation[1], -viewTranslation[2]);
  glGetFloatv(GL_MODELVIEW_MATRIX, modelView);
  glPopMatrix();
  for(int i=0; i<12; i++) invViewMatrix[i] = modelView[4*(i&3)+i/4];
  if(CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeof(float4)*3))) throw(-1);
  //copyInvViewMatrix(invViewMatrix, sizeof(float4)*3);
  imHost.bmp = im2D.map4draw();
  imHost.bind2draw();
  if(CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(im), &imHost, sizeof(imHost)))) throw(-1);
  if(CHECK_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(im3D), this, sizeof(im3D_pars)))) throw(-1);
  //if(CHECK_ERROR(hipDeviceSetCacheConfig(hipFuncCachePreferShared))) throw(-1);
  switch(mode3D) {
    case 0: render3D <<<dim3(bNx/NW,bNy/NW),dim3(NW,NW)>>>(); break;
    case 1:
#ifdef SURF
      surf_render3D <<<dim3(bNx/NW,bNy/NW),dim3(NW,NW)>>>();
#else//SURF
      printf("Для задействования визуализации на поверхности скомпилируйте im3D.cu с опцией -DSURF или используйте im3Dsurf\n");
#endif//SURF
      break;
    case 2: grad_render3D <<<dim3(bNx/NW,bNy/NW),dim3(NW,NW)>>>(); break;
  }
  if(imHost.draw_flag) {
    //if(mode3D<=1) 
      draw_pal <<<bNx/NW,NW>>>();
    if(mode3D>0) draw_pal3D <<<NW,NW>>>();
    //if(imHost.palDim <= 2) draw_pal <<<bNx/NW,NW>>>();
    //else if(imHost.palDim > 1) draw_pal3D <<<NW,NW>>>();
  } else draw_wavelength_pal <<<bNx/NW,NW>>>();
  if(imHost.negate_flag) negate <<<bNx/NW,NW>>>();
  imHost.nFrame++;
  imHost.unbindAfterDraw();
  im2D.unmapAfterDraw();
} catch(...) {
  printf("recalc3D_im3D: Возникла какая-то ошибка.\n");
}
}

#include <hipfft/hipfft.h>

//inline __device__ float my_fabsC(float2& v) { return v.x;}//hypotf(v.x, v.y); }
inline __device__ float my_fabsC(float2& v) { return hypotf(v.x, v.y); }
inline __device__ int my_abs(int v) { return v>=0?v:-v; }
//inline __device__ int my_abs(int v) { return v==0?1:v>=0?v:-v; }

__global__ void cmplx2abs(hipfftComplex *dataC, hipfftReal *dataR) {
  //float* pC=(float*)(dataC+blockIdx.x*(blockDim.x/2+1));
  //dataR[blockIdx.x*blockDim.x+threadIdx.x] = pC[threadIdx.x];
  dataR[blockIdx.x*blockDim.x+threadIdx.x] = my_fabsC(dataC[blockIdx.x*(blockDim.x/2+1)+my_abs(blockDim.x/2-threadIdx.x)]);
}
#define CHECK_ERROR_FFT(err) CheckErrorFFT( err, __FILE__,__LINE__)
bool CheckErrorFFT(hipfftResult rs, const char *file, int line) {
  if(rs == HIPFFT_SUCCESS) return false;
  const char* err="Непонятная ошибка в cuFFT";
  switch(rs) {
  case HIPFFT_SUCCESS: err = "0, // The cuFFT operation was successful";
  case HIPFFT_INVALID_PLAN: err = "1, // cuFFT was passed an invalid plan handle";
  case HIPFFT_ALLOC_FAILED: err = "2, // cuFFT failed to allocate GPU or CPU memory";
  case HIPFFT_INVALID_TYPE: err = "3, // No longer used";
  case HIPFFT_INVALID_VALUE: err = "4, // User specified an invalid pointer or parameter";
  case HIPFFT_INTERNAL_ERROR: err = "5, // Driver or internal cuFFT library error";
  case HIPFFT_EXEC_FAILED: err = "6, // Failed to execute an FFT on the GPU";
  case HIPFFT_SETUP_FAILED: err = "7, // The cuFFT library failed to initialize";
  case HIPFFT_INVALID_SIZE: err = "8, // User specified an invalid transform size";
  case HIPFFT_UNALIGNED_DATA: err = "9, // No longer used";
  case HIPFFT_INCOMPLETE_PARAMETER_LIST: err = "10, // Missing parameters in call";
  case HIPFFT_INVALID_DEVICE: err = "11, // Execution of a plan was on different GPU than plan creation";
  case HIPFFT_PARSE_ERROR: err = "12, // Internal plan database error";
  case HIPFFT_NO_WORKSPACE: err = "13 // No workspace has been provided prior to plan execution";
  };
  fprintf(stderr, "%s in %s at line %d\n", err, file, line);
  return true;
}
void makeFFTz(float* buf, int Nx, int Ny, int Nz) {
try {
  hipfftHandle plan;
  hipfftComplex *dataC; hipfftReal *dataR;
  if(CHECK_ERROR(hipMalloc((void**)&dataC, sizeof(hipfftComplex)*(Nz/2+1)*Nx*Ny))) throw(-1);
  if(CHECK_ERROR(hipMalloc((void**)&dataR, sizeof(hipfftReal)*Nz*Nx*Ny))) throw(-1);
  if(CHECK_ERROR(hipMemcpy(dataR, buf, 4*Nz*Nx*Ny, hipMemcpyHostToDevice))) throw(-1);
  if(CHECK_ERROR_FFT(hipfftPlan1d(&plan, Nz, HIPFFT_R2C, Nx*Ny))) throw(-1);
  if(CHECK_ERROR_FFT(hipfftExecR2C(plan, dataR, dataC))) throw(-1);
  if(CHECK_ERROR(hipDeviceSynchronize())) throw(-1);
  cmplx2abs <<<Nx*Ny,Nz>>>(dataC, dataR);
  if(CHECK_ERROR(hipDeviceSynchronize())) throw(-1);
  if(CHECK_ERROR(hipMemcpy(buf, dataR, 4*Nz*Nx*Ny, hipMemcpyDeviceToHost))) throw(-1);
  if(CHECK_ERROR_FFT(hipfftDestroy(plan))) throw(-1);
  if(CHECK_ERROR(hipFree(dataC))) throw(-1);
  if(CHECK_ERROR(hipFree(dataR))) throw(-1);
} catch(...) {
  printf("Ошибка в makeFFTz.\n");
}
}
void im3D_pars::initCuda(Arr3D_pars& arr) {
    //printf("==============\n");
    //for(int ix=0; ix<Nx; ix++) for(int iy=0; iy<Ny; iy++) for(int iz=0; iz<Nz; iz++) arr.Arr3Dbuf[iz*Ny*Nx+iy*Nx+ix]=exp(-0.01*ix);
  // create transfer function texture
  //hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
  //if(CHECK_ERROR(hipMalloc3DArray(&data3D_texArray, &channelDesc, make_hipExtent(Nx,Ny,Nz)))) throw(-1);
  hipMemcpy3DParms myparms={0};
  myparms.srcPos = make_hipPos(0,0,0);
  myparms.dstPos = make_hipPos(0,0,0);
  myparms.srcPtr = make_hipPitchedPtr(arr.Arr3Dbuf, Nx*sizeof(floatT4im), Nx, Ny);
  myparms.dstArray = data3D_texArray;
  myparms.extent = make_hipExtent(Nx,Ny,Nz);
  myparms.kind = arr.inGPUmem?hipMemcpyDeviceToDevice:hipMemcpyHostToDevice;
  if(CHECK_ERROR(hipMemcpy3D(&myparms))) throw(-1);
  //if(draw_edges_flag) draw_edges(imHost.fmax);
  initTex();
}
void im3D_pars::initTex() {
  data3D_tex.normalized = false;//true;
  data3D_tex.filterMode = ((render_type==3)==filterMode_flag)?hipFilterModeLinear:hipFilterModePoint; //Point;//filter_pal?hipFilterModePoint:hipFilterModeLinear;
  data3D_tex.addressMode[0] = hipAddressModeClamp;//cyclic_pal?hipAddressModeWrap:hipAddressModeClamp;
  data3D_tex.addressMode[1] = hipAddressModeClamp;//cyclic_pal?hipAddressModeWrap:hipAddressModeClamp;
  data3D_tex.addressMode[2] = hipAddressModeClamp;//cyclic_pal?hipAddressModeWrap:hipAddressModeClamp;
  if(CHECK_ERROR(hipBindTextureToArray(data3D_tex, data3D_texArray))) throw(-1);
}
void im3D_pars::initCuda_surf(Arr3D_pars& arr, size_t sh) {
#ifdef SURF
  hipMemcpy3DParms myparms={0};
  myparms.srcPos = make_hipPos(0,0,0);
  myparms.dstPos = make_hipPos(0,0,0);
  size_t N=Nx; N*=Ny; N*=Nz;
  myparms.srcPtr = make_hipPitchedPtr(arr.Arr3Dbuf+sh, Nx*sizeof(short2), Nx, Ny);
  myparms.dstArray = data3Dsurf_texArray;
  myparms.extent = make_hipExtent(Nx,Ny,Nz);
  myparms.kind = arr.inGPUmem?hipMemcpyDeviceToDevice:hipMemcpyHostToDevice;
  if(CHECK_ERROR(hipMemcpy3D(&myparms))) throw(-1);
  initTex_surf();
#endif//SURF
}
void im3D_pars::initTex_surf() {
  data3Dsurf_tex.normalized = false;//true;
  data3Dsurf_tex.filterMode = hipFilterModePoint; //Point;//filter_pal?hipFilterModePoint:hipFilterModeLinear;
  data3Dsurf_tex.addressMode[0] = hipAddressModeClamp;//cyclic_pal?hipAddressModeWrap:hipAddressModeClamp;
  data3Dsurf_tex.addressMode[1] = hipAddressModeClamp;//cyclic_pal?hipAddressModeWrap:hipAddressModeClamp;
  data3Dsurf_tex.addressMode[2] = hipAddressModeClamp;//cyclic_pal?hipAddressModeWrap:hipAddressModeClamp;
  if(CHECK_ERROR(hipBindTextureToArray(data3Dsurf_tex, data3Dsurf_texArray))) throw(-1);
}
void reset(im3D_pars* p) {
  imHost.reset();
  imHost.set_lim(-1.f,1.f);
  imHost.draw_flag = imHost.negate_flag = imHost.centric_pal = true;
  imHost.cyclic_pal = false;
  if(p) p->reset();
}
void im3D_pars::init3D(Arr3D_pars& arr) {
  //::reset();
  optfid = open(optfName, O_RDWR|O_CREAT, 0644);
  if(optfid<0) printf("Не могу открыть файл %s, сохранение/загрузка наборов опций визуализации невозможна\n", optfName);
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<floatT4im>();
  printf("im3D_pars::init3D: Nx,Ny,Nz=%d,%d,%d\n", Nx,Ny,Nz);
  if(CHECK_ERROR(hipMalloc3DArray(&data3D_texArray, &channelDesc, make_hipExtent(Nx,Ny,Nz)))) throw(-1);
  if(CHECK_ERROR(hipMalloc(&randArr, NW*NW*sizeof(float)))) throw(-1);
  hiprandState *devStates;
  hipMalloc( (void **)&devStates, NW*NW*sizeof(hiprandState) );
  init_rand<<<NW,NW>>>(devStates,randArr);
  if(CHECK_ERROR(hipDeviceSynchronize())) throw(-1);
  hipFree(devStates);

  //initCuda(arr); ---- !!!!!!!!!!!!!!!!!!!
#ifdef SURF
  if(CHECK_ERROR(hipDeviceSynchronize())) throw(-1);
  channelDesc = hipCreateChannelDesc<short2>();
  if(CHECK_ERROR(hipMalloc3DArray(&data3Dsurf_texArray, &channelDesc, make_hipExtent(Nx,Ny,Nz)))) throw(-1);
  if(CHECK_ERROR(hipDeviceSynchronize())) throw(-1);
  //initCuda_surf(arr); ----- !!!!!!!!!!!!!!!!!!1
#endif//SURF
}
void im3D_pars::recalc_func() {
  if(recalc_always || recalc_at_once) {
    if(recalc_at_once) recalc_at_once=false;
    else xyz->step();
    cudaTimer tm; tm.start();
    if(draw_bmp4backgrownd>=2 && render_type==3) {
      switch(draw_bmp4backgrownd) {
      case 2: recalc_im3D(); break;
      case 3: recalc_sec_im3D(); break;
      }
      save_bmp4backgrownd();
    }
    switch(render_type) {
    case 2: recalc_im3D(); break;
    case 3: recalc3D_im3D(); break;
    }
    runTime=tm.stop(); SmoothFPS = 0.9*SmoothFPS+100./runTime;
    if(type_diag_flag>=2) printf("Frame %d (%.2f/%.2f fps), last run Times: %7.2f msec\n", imHost.nFrame, SmoothFPS, 1000./runTime, runTime);
  }
}
int im3D_pars::init_from_command_line(char** argv) {
  if(strcmp(*argv,"--sensor")==0) { float v[3]; read_float3(v, argv[1]); icalcNdrop.add_sensor(v[0], v[1], v[2]); return 2; }
  return im3D_pars4save::init_from_command_line(argv);
}
floatT4im Arr3D_pars::get_val_from_arr3D(int ix, int iy, int iz) {
  if(inCPUmem) return ((floatT4im*)Arr3Dbuf)[get_ind(ix,iy,iz)];
  floatT4im res;
  if(inGPUmem) CHECK_ERROR(hipMemcpy(&res, get_ptr((sizeof(floatT4im)/sizeof(float))*ix,iy,iz), sizeof(floatT4im), hipMemcpyDeviceToHost));
  return res;
}
/*
__global__ void calc_limits(float* buf, float* fLims, int Nxv, int Nxa, int Nxs) {
  float2 fLim;
  float* pf=buf+blockIdx.x*Nxv+threadIdx.x;
  fLim.x = fLim.y = *pf;

  for(int i=0; i<Nxs; i++,pf+=Nxa*Nxv) {
    float v=*pf;
    if(v<fLim.x) fLim.x = v;
    if(v>fLim.y) fLim.y = v;
  }
  __shared__ float2 fLim_sh[Nxv];
  fLim_sh[threadIdx.x] = fLim;
  __syncthreads();
  if(threadIdx.x>warpSize) return;
  for(int i=threadIdx.x; i<Nxv; i+=warpSize) {
    float2 v=fLim_sh[i];
    if(v.x<fLim.x) fLim.x = v.x;
    if(v.y>fLim.y) fLim.y = v.y;
  }
  fLim_sh[threadIdx.x] = fLim;
  if(threadIdx.x>0) return;
  for(int i=0; i<warpSize; i++) {
    float2 v=fLim_sh[i];
    if(v.x<fLim.x) fLim.x = v.x;
    if(v.y>fLim.y) fLim.y = v.y;
  }
  fLims[2*blockIdx.x  ] = fLim.x;
  fLims[2*blockIdx.x+1] = fLim.y;
}

void Arr3D_pars::set_lim_from_arr3D() {
  if(inCPUmem) reset_min_max();
  if(inGPUmem) {
    float* fLims=0,* fLimsD=0;
    CHECK_ERROR(hipMalloc((void**) &fLimsD, 2*Ny*sizeof(float)));
    calc_limits<<<Ny,Nx>>>(Arr3Dbuf, fLimsD, Nx, Ny, Nz);
    fLims=new float[2*Ny];
    CHECK_ERROR(hipMemcpy(fLims, fLimsD, 2*Ny*sizeof(float), hipMemcpyDeviceToHost));
    CHECK_ERROR(hipFree(fLimsD));
    fMin = fLims[0]; fMax = fLims[1];
    for(int i=0; i<Ny; i++) {
      if(fLims[2*i  ]<fMin) fMin = fLims[2*i  ];
      if(fLims[2*i+1]>fMax) fMax = fLims[2*i+1];
    }
    delete fLims;
  }
}*/
