#include "hip/hip_runtime.h"
#include "structs.cuh"
#include "init.h"
#include "LBMconsts.cuh"
#include "phys.h"
#include <nvfunctional>

#include "materials.cuh"

template<class F> __global__ void fill(F);

void init(){
  parsHost.iStep=0;
  parsHost.StepsMax=PPhost.MaxSteps;
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );

  printf("Malloc data\n");
  parsHost.data.malloc_data(Nx,Ny,Nz);
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );

  cuTimer init_timer;
  //fill<<<dim3(Nx,Ny),Nz>>>( [] __device__(int ix, int iy,int iz) {return blank_mat(ix,iy,iz);} );
  fill<<<dim3(Nx,Ny),Nz>>>( [] __device__(int ix, int iy,int iz) {
     //return sinTemperature(ix,iy,iz);
     return shear_wave(ix,iy,iz);
     //return vortex_mat(ix,iy,iz);
     //return TGV_mat(ix,iy,iz);
     //return blank_mat(ix,iy,iz);
  } );
  hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  printf("\n");
  printf("TLat=%16.9g\n\n",LBMconsts::TLat);

  printf("Initialization time: %.2f ms\n", init_timer.gettime());
  
  copy2dev( parsHost, pars );
  copy2dev( PPhost, PPdev );

}

template<class F> __global__ void fill(F func){
  /*int ix = threadIdx.x+blockIdx.x*blockDim.x;
  int iy = threadIdx.y+blockIdx.y*blockDim.y;
  int iz = threadIdx.z+blockIdx.z*blockDim.z;*/
  int ix = blockIdx.x;
  int iy = blockIdx.y;
  int iz = threadIdx.x;
  Cell c;
  std::pair<ftype, ftype4> rho_uT = func(ix,iy,iz);

  const ftype rho = rho_uT.first;
  const ftype3 vel = make_ftype3(rho_uT.second.x, rho_uT.second.y, rho_uT.second.z) ;
  const ftype T = rho_uT.second.w;

  assert(rho_uT.second.w>LBMconsts::Tmin);
  assert(rho_uT.second.w<LBMconsts::Tmax);

  ftype feq[LBMconsts::Qn];
  #ifdef SEMI_LAGRANGIAN
  c.calcEq(feq, rho_uT.first, make_ftype3(0,0,0), LBMconsts::cs2 );
  #else
  c.calcEq(feq, rho_uT.first, vel, rho_uT.second.w );
  #endif
  for(int iq=0; iq<LBMconsts::Qn; iq++) c.f[iq]=feq[iq];
  c.rho = rho;
  c.vel = vel;
  c.T = T;
  for(int iq=0; iq<LBMconsts::Qn; iq++) {
    if(c.f[iq]<=0) printf("Warning CELL (%4d %4d %4d): f[%d] is negative\n", ix,iy,iz, iq);
  }

  pars.data.set_cell(c, 0, ix,iy,iz);
  pars.data.set_cell(c, 1, ix,iy,iz);

  //using namespace LBMconsts;
  //if(ix==0 && iy==0 && iz==0) printf("TLAT = %g W0123 = %g %g %g %g\n", TLat, W0get(TLat), W1get(TLat), W2get(TLat), W3get(TLat));

}

void PhysPars::setupUnits(){
  RhoUnitConv = 1;
  VelUnitConv = dt/dr;
  ViscUnitConv = dt/(dr*dr);
  const ftype ViscAtTUnitConv = 1./dt;
  tau = 0.5+visc_atT*ViscAtTUnitConv;
  dtau = 1/tau;
  TempUnitConv = dt*dt/(dr*dr);
  printf("Density Units Converstion Coeff = %g\n", RhoUnitConv);
  printf("Velocity Units Converstion Coeff = %g\n", VelUnitConv);
  printf("Temperature Units Converstion Coeff = %g\n", TempUnitConv);
  printf("Viscosity Units Converstion Coeff = %g\n", ViscUnitConv);
  printf("Tau relaxation = %g\n", tau);
}
